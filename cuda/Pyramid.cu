#include <cfloat>

#include "Error.hpp"
#include "Pyramid.hpp"
#include "Kernels.hpp"

void Pyramid::intersect(float *d_rx, float *d_ry, float *d_rz,
                        float *d_vx, float *d_vy, float *d_vz,
                        const int N, const int blockSize, const int numBlocks,
                        std::vector<float> &int_times, std::vector<float> &int_coords)
{
    float *device_time;
    CudaErrchk( hipMalloc(&device_time, 5*N*sizeof(float)) );
    initArray<<<numBlocks, blockSize>>>(device_time, 5*N, -5);
    CudaErrchkNoCode();
    float *intersect;
    CudaErrchk( hipMalloc(&intersect, 6*N*sizeof(float)) );
    initArray<<<numBlocks, blockSize>>>(intersect, 6*N, FLT_MAX);
    CudaErrchkNoCode();
    float *simp_times;
    CudaErrchk( hipMalloc(&simp_times, 2*N*sizeof(float)) );
    initArray<<<numBlocks, blockSize>>>(simp_times, 2*N, -5);
    CudaErrchkNoCode();
    int_times.resize(2*N);
    int_coords.resize(6*N);
    intersectPyramid<<<numBlocks, blockSize>>>(d_rx, d_ry, d_rz,
                                               d_vx, d_vy, d_vz,
                                               edgeX, edgeY, height,
                                               N, device_time, intersect);
    //CudaDeviceSynchronize();
    //printf("\n\nEnd Kernel.\n");
    /*std::vector<float> tmp;
    tmp.resize(5*N);
    CudaErrchk( hipMemcpy(tmp.data(), device_time, 5*N*sizeof(float), hipMemcpyDeviceToHost) );
    for (int i = 0; i < (int)(tmp.size()); i++)
    {
        if (i % 5 == 0)
        {
            printf("Ray Index %i:\n", i/5);
        }
        printf("    Offset = %i: Time = %f\n", (i%5), tmp[i]);
    }*/
    hipDeviceSynchronize();
    simplifyTimes<<<numBlocks, blockSize>>>(device_time, N, 5, simp_times);
    CudaErrchkNoCode();
    float *it = int_times.data();
    float *ic = int_coords.data();
    CudaErrchk( hipMemcpy(it, simp_times, 2*N*sizeof(float), hipMemcpyDeviceToHost) );
    CudaErrchk( hipMemcpy(ic, intersect, 6*N*sizeof(float), hipMemcpyDeviceToHost) );
    CudaErrchk( hipFree(device_time) );
    CudaErrchk( hipFree(intersect) );
    CudaErrchk( hipFree(simp_times) );
}
