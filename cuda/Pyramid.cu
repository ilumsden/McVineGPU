#include <cfloat>

#include "Error.hpp"
#include "Pyramid.hpp"

namespace mcvine
{

    namespace gpu
    {

        namespace composite
        {

            void Pyramid::exteriorIntersect(Vec3<float> *d_origins, Vec3<float> *d_vel,
                                            const int N, const int blockSize, const int numBlocks,
                                            std::vector<float> &int_times, 
                                            std::vector< Vec3<float> > &int_coords)
            {
                namespace kernels = mcvine::gpu::kernels;
                /* The device float array "device_time" is allocated on device, and
                 * its elements' values are set to -5.
                 * This array will store the times calculated by the intersectPyramid
                 * kernel.
                 */
                float *device_time;
                CudaErrchk( hipMalloc(&device_time, 5*N*sizeof(float)) );
                kernels::initArray<float><<<numBlocks, blockSize>>>(device_time, 5*N, -5);
                CudaErrchkNoCode();
                /* The device Vec3<float> array "intersect" is allocated on device, and
                 * its elements' values are set to FLT_MAX.
                 * This array will store the intersection coordinates calculated
                 * by the intersectPyramid kernel.
                 */
                Vec3<float> *d_intersect;
                CudaErrchk( hipMalloc(&d_intersect, 2*N*sizeof(Vec3<float>)) );
                kernels::initArray< Vec3<float> ><<<numBlocks, blockSize>>>(d_intersect, 2*N, Vec3<float>(FLT_MAX, FLT_MAX, FLT_MAX));
                CudaErrchkNoCode();
                /* The device float array "simp_times" is allocated on device, and
                 * its elements' values are set to -5.
                 * This array will store the output of the simplifyTimes kernel.
                 */
                float *simp_times;
                CudaErrchk( hipMalloc(&simp_times, 2*N*sizeof(float)) );
                kernels::initArray<float><<<numBlocks, blockSize>>>(simp_times, 2*N, -5);
                CudaErrchkNoCode();
                float *d_data;
                CudaErrchk( hipMalloc(&d_data, 3*sizeof(float)) );
                CudaErrchk( hipMemcpy(d_data, data, 3*sizeof(float), hipMemcpyHostToDevice) );
                // These vectors are resized to match the size of the arrays above.
                int_times.resize(2*N);
                int_coords.resize(2*N);
                // The kernels are called to perform the intersection calculation.
                kernels::intersect<<<numBlocks, blockSize>>>(interKeyDict[type],
                                                             d_origins, d_vel, d_data, N,
                                                             device_time, d_intersect);
                kernels::simplifyTimePointPairs<<<numBlocks, blockSize>>>(
                    device_time,
                    d_intersect,
                    N, 5, 2, 2,
                    simp_times,
                    d_intersect);
                kernels::forceIntersectionOrder<<<numBlocks, blockSize>>>(simp_times, d_intersect, N);
                CudaErrchkNoCode();
                /* The data from simp_times and intersect is copied into
                 * int_times and int_coords respectively.
                 */
                float *it = int_times.data();
                Vec3<float> *ic = int_coords.data();
                CudaErrchk( hipMemcpy(it, simp_times, 2*N*sizeof(float), hipMemcpyDeviceToHost) );
                CudaErrchk( hipMemcpy(ic, d_intersect, 2*N*sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
                /* The device memory allocated at the beginning of the function
                 * is freed.
                 */
                CudaErrchk( hipFree(device_time) );
                CudaErrchk( hipFree(d_intersect) );
                CudaErrchk( hipFree(simp_times) );
                CudaErrchk( hipFree(d_data) );
            }

            void Pyramid::interiorIntersect(Vec3<float> *d_origins, Vec3<float> *d_vel,
                                            const int N, const int blockSize, const int numBlocks,
                                            std::vector<float> &int_times, 
                                            std::vector< Vec3<float> > &int_coords)
            {
                namespace kernels = mcvine::gpu::kernels;
                /* The device float array "device_time" is allocated on device, and
                 * its elements' values are set to -5.
                 * This array will store the times calculated by the intersectPyramid
                 * kernel.
                 */
                float *device_time;
                CudaErrchk( hipMalloc(&device_time, 5*N*sizeof(float)) );
                kernels::initArray<float><<<numBlocks, blockSize>>>(device_time, 5*N, -5);
                CudaErrchkNoCode();
                /* The device Vec3<float> array "intersect" is allocated on device, and
                 * its elements' values are set to FLT_MAX.
                 * This array will store the intersection coordinates calculated
                 * by the intersectPyramid kernel.
                 */
                Vec3<float> *d_intersect;
                CudaErrchk( hipMalloc(&d_intersect, 2*N*sizeof(Vec3<float>)) );
                kernels::initArray< Vec3<float> ><<<numBlocks, blockSize>>>(d_intersect, 2*N, Vec3<float>(FLT_MAX, FLT_MAX, FLT_MAX));
                CudaErrchkNoCode();
                /* The device float array "simp_times" is allocated on device, and
                 * its elements' values are set to -5.
                 * This array will store the output of the simplifyTimes kernel.
                 */
                float *simp_times;
                CudaErrchk( hipMalloc(&simp_times, N*sizeof(float)) );
                kernels::initArray<float><<<numBlocks, blockSize>>>(simp_times, N, -5);
                CudaErrchkNoCode();
                /* The Vec3<float> array "simp_times" is allocated on device, and
                 * its elements' values are set to FLT_MAX.
                 * This array will store the output of the simplifyPoints kernel.
                 */
                Vec3<float> *simp_int;
                CudaErrchk( hipMalloc(&simp_int, N*sizeof(Vec3<float>)) );
                kernels::initArray< Vec3<float> ><<<numBlocks, blockSize>>>(simp_int, N, Vec3<float>(FLT_MAX, FLT_MAX, FLT_MAX));
                CudaErrchkNoCode();
                float *d_data;
                CudaErrchk( hipMalloc(&d_data, 3*sizeof(float)) );
                CudaErrchk( hipMemcpy(d_data, data, 3*sizeof(float), hipMemcpyHostToDevice) );
                // These vectors are resized to match the size of the arrays above.
                int_times.resize(N);
                int_coords.resize(N);
                // The kernels are called to perform the intersection calculation.
                /*intersectPyramid<<<numBlocks, blockSize>>>(d_origins, d_vel,
                                                           edgeX, edgeY, height,
                                                           N, device_time, intersect);*/
                kernels::intersect<<<numBlocks, blockSize>>>(interKeyDict[type],
                                                             d_origins, d_vel, d_data, N,
                                                             device_time, d_intersect);
                kernels::simplifyTimePointPairs<<<numBlocks, blockSize>>>(
                    device_time,
                    d_intersect,
                    N, 5, 2, 1,
                    simp_times,
                    simp_int);
                CudaErrchkNoCode();
                /* The data from simp_times and intersect is copied into
                 * int_times and int_coords respectively.
                 */
                float *it = int_times.data();
                Vec3<float> *ic = int_coords.data();
                CudaErrchk( hipMemcpy(it, simp_times, N*sizeof(float), hipMemcpyDeviceToHost) );
                CudaErrchk( hipMemcpy(ic, simp_int, N*sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
                /* The device memory allocated at the beginning of the function
                 * is freed.
                 */
                CudaErrchk( hipFree(device_time) );
                CudaErrchk( hipFree(d_intersect) );
                CudaErrchk( hipFree(simp_times) );
                CudaErrchk( hipFree(simp_int) );
                CudaErrchk( hipFree(d_data) );
            }

        }

    }

}
