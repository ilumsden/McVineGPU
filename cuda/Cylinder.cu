#include "Cylinder.hpp"
#include "Error.hpp"

void Cylinder::intersect(float *d_rx, float *d_ry, float *d_rz,
                         float *d_vx, float *d_vy, float *d_vz,
                         const int N, const int blockSize, const int numBlocks,
                         std::vector<float> &int_times, std::vector<float> &int_coords)
{
    float *device_time;
    CudaErrchk( hipMalloc(&device_time, 4*N*sizeof(float)) );
    initArray<<<numBlocks, blockSize>>>(device_time, 4*N, -5);
    CudaErrchkNoCode();
    float *intersect;
    CudaErrchk( hipMalloc(&intersect, 6*N*sizeof(float)) );
    initArray<<<numBlocks, blockSize>>>(intersect, 6*N, FLT_MAX);
    CudaErrchkNoCode();
    float *simp_times;
    CudaErrchk( hipMalloc(&simp_times, 2*N*sizeof(float)) );
    initArray<<<numBlocks, blockSize>>>(simp_times, 2*N, -5);
    CudaErrchkNoCode();
    int_times.resize(2*N);
    int_coords.resize(6*N);
    intersectCylinder<<<numBlocks, blockSize>>>(d_rx, d_ry, d_rz,
                                                d_vx, d_vy, d_vz,
                                                radius, height,
                                                N, device_time, intersect);
    simplifyTimes<<<numBlocks, blockSize>>>(device_time, N, 4, simp_times);
    CudaErrchkNoCode();
    float *it = int_times.data();
    float *ic = int_coords.data();
    CudaErrchk( hipMemcpy(it, simp_times, 2*N*sizeof(float), hipMemcpyDeviceToHost) );
    CudaErrchk( hipMemcpy(ic, intersect, 6*N*sizeof(float), hipMemcpyDeviceToHost) );
    CudaErrchk( hipFree(device_time) );
    CudaErrchk( hipFree(intersect) );
    CudaErrchk( hipFree(simp_times) );
}
