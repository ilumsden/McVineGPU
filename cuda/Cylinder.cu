#include "Cylinder.hpp"
#include "Error.hpp"

void Cylinder::exteriorIntersect(Vec3<float> *d_origins, Vec3<float> *d_vel,
                                 const int N, const int blockSize, const int numBlocks,
                                 std::vector<float> &int_times, 
                                 std::vector< Vec3<float> > &int_coords)
{
    /* The device float array "device_time" is allocated on device, and
     * its elements' values are set to -5.
     * This array will store the times calculated by the intersectCylinder
     * kernel.
     */
    float *device_time;
    CudaErrchk( hipMalloc(&device_time, 4*N*sizeof(float)) );
    initArray<float><<<numBlocks, blockSize>>>(device_time, 4*N, -5);
    CudaErrchkNoCode();
    /* The device Vec3<float> array "intersect" is allocated on the device, and
     * its elements' values are set to FLT_MAX.
     * This array will store the intersection coordinates calculated
     * by the intersectCylinder kernel.
     */
    Vec3<float> *d_intersect;
    CudaErrchk( hipMalloc(&d_intersect, 2*N*sizeof(Vec3<float>)) );
    initArray< Vec3<float> ><<<numBlocks, blockSize>>>(d_intersect, 2*N, Vec3<float>(FLT_MAX, FLT_MAX, FLT_MAX));
    CudaErrchkNoCode();
    /* The device float array "simp_times" is allocated on device, and
     * its elements' values are set to -5.
     * This array will store the output of the simplifyTimes kernel.
     */
    float *simp_times;
    CudaErrchk( hipMalloc(&simp_times, 2*N*sizeof(float)) );
    initArray<float><<<numBlocks, blockSize>>>(simp_times, 2*N, -5);
    CudaErrchkNoCode();
    float *d_data;
    CudaErrchk( hipMalloc(&d_data, 2*sizeof(float)) );
    CudaErrchk( hipMemcpy(d_data, data, 2*sizeof(float), hipMemcpyHostToDevice) );
    // These vectors are resized to match the size of the arrays above.
    int_times.resize(2*N);
    int_coords.resize(2*N);
    // The kernels are called to perform the intersection calculation.
    /*intersectCylinder<<<numBlocks, blockSize>>>(d_origins, d_vel,
                                                radius, height,
                                                N, device_time, intersect);*/
    intersect<<<numBlocks, blockSize>>>(interKeyDict[type],
                                        d_origins, d_vel, d_data, N,
                                        device_time, d_intersect);
    simplifyTimePointPairs<<<numBlocks, blockSize>>>(device_time,
                                                     d_intersect,
                                                     N, 4, 2, 2,
                                                     simp_times,
                                                     d_intersect);
    forceIntersectionOrder<<<numBlocks, blockSize>>>(simp_times, d_intersect, N);
    CudaErrchkNoCode();
    /* The data from simp_times and intersect is copied into
     * int_times and int_coords respectively.
     */
    float *it = int_times.data();
    Vec3<float> *ic = int_coords.data();
    CudaErrchk( hipMemcpy(it, simp_times, 2*N*sizeof(float), hipMemcpyDeviceToHost) );
    CudaErrchk( hipMemcpy(ic, d_intersect, 2*N*sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
    /* The device memory allocated at the beginning of the function
     * is freed.
     */
    CudaErrchk( hipFree(device_time) );
    CudaErrchk( hipFree(d_intersect) );
    CudaErrchk( hipFree(simp_times) );
    CudaErrchk( hipFree(d_data) );
}

void Cylinder::interiorIntersect(Vec3<float> *d_origins, Vec3<float> *d_vel,
                                 const int N, const int blockSize, const int numBlocks,
                                 std::vector<float> &int_times, 
                                 std::vector< Vec3<float> > &int_coords)
{
    /* The device float array "device_time" is allocated on device, and
     * its elements' values are set to -5.
     * This array will store the times calculated by the intersectCylinder
     * kernel.
     */
    float *device_time;
    CudaErrchk( hipMalloc(&device_time, 4*N*sizeof(float)) );
    initArray<float><<<numBlocks, blockSize>>>(device_time, 4*N, -5);
    CudaErrchkNoCode();
    /* The device Vec3<float> array "intersect" is allocated on the device, and
     * its elements' values are set to FLT_MAX.
     * This array will store the intersection coordinates calculated
     * by the intersectCylinder kernel.
     */
    Vec3<float> *d_intersect;
    CudaErrchk( hipMalloc(&d_intersect, 2*N*sizeof(Vec3<float>)) );
    initArray< Vec3<float> ><<<numBlocks, blockSize>>>(d_intersect, 2*N, Vec3<float>(FLT_MAX, FLT_MAX, FLT_MAX));
    CudaErrchkNoCode();
    /* The device float array "simp_times" is allocated on device, and
     * its elements' values are set to -5.
     * This array will store the output of the simplifyTimes kernel.
     */
    float *simp_times;
    CudaErrchk( hipMalloc(&simp_times, N*sizeof(float)) );
    initArray<float><<<numBlocks, blockSize>>>(simp_times, N, -5);
    CudaErrchkNoCode();
    /* The Vec3<float> array "simp_times" is allocated on device, and
     * its elements' values are set to FLT_MAX.
     * This array will store the output of the simplifyPoints kernel.
     */
    Vec3<float> *simp_int;
    CudaErrchk( hipMalloc(&simp_int, N*sizeof(Vec3<float>)) );
    initArray< Vec3<float> ><<<numBlocks, blockSize>>>(simp_int, N, Vec3<float>(FLT_MAX, FLT_MAX, FLT_MAX));
    CudaErrchkNoCode();
    float *d_data;
    CudaErrchk( hipMalloc(&d_data, 2*sizeof(float)) );
    CudaErrchk( hipMemcpy(d_data, data, 2*sizeof(float), hipMemcpyHostToDevice) );
    // These vectors are resized to match the size of the arrays above.
    int_times.resize(N);
    int_coords.resize(N);
    // The kernels are called to perform the intersection calculation.
    /*intersectCylinder<<<numBlocks, blockSize>>>(d_origins, d_vel,
                                                radius, height,
                                                N, device_time, intersect);*/
    intersect<<<numBlocks, blockSize>>>(interKeyDict[type],
                                        d_origins, d_vel, d_data, N,
                                        device_time, d_intersect);
    simplifyTimePointPairs<<<numBlocks, blockSize>>>(device_time,
                                                     d_intersect,
                                                     N, 4, 2, 1,
                                                     simp_times,
                                                     simp_int);
    CudaErrchkNoCode();
    /* The data from simp_times and intersect is copied into
     * int_times and int_coords respectively.
     */
    float *it = int_times.data();
    Vec3<float> *ic = int_coords.data();
    CudaErrchk( hipMemcpy(it, simp_times, N*sizeof(float), hipMemcpyDeviceToHost) );
    CudaErrchk( hipMemcpy(ic, simp_int, N*sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
    /* The device memory allocated at the beginning of the function
     * is freed.
     */
    CudaErrchk( hipFree(device_time) );
    CudaErrchk( hipFree(d_intersect) );
    CudaErrchk( hipFree(simp_times) );
    CudaErrchk( hipFree(simp_int) );
    CudaErrchk( hipFree(d_data) );
}
