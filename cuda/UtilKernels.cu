#include "hip/hip_runtime.h"
#include <cstdio>

#include "UtilKernels.hpp"

__device__ bool solveQuadratic(float a, float b, float c, float &x0, float &x1)
{
    // Calculates the discriminant and returns false if it is less than 0.
    float discr = b*b - 4*a*c;
    if (discr < 0)
    {
        return false;
    }
    else
    {
        /* This process ensures that there is little to no roundoff error
         * in the evaluation of the quadratic formula.
         * This process defines a value 
         * q = -0.5 * (b + sign(b)*sqrt(b^2 - 4ac)).
         * If you define x0 = q/a (producing the standard quadratic
         * formula), x1 can be defined as c/q by multiplying the
         * other form of the formula (+/- -> -sign(b)) by
         * ((-b + sign(b)*sqrt(discr))/(-b + sign(b)*sqrt(discr))).
         */
        float q = (b > 0) ? 
                  (-0.5 * (b + sqrtf(discr))) :
                  (-0.5 * (b - sqrtf(discr)));
        x0 = q/a;
        x1 = c/q;
    }
    // This simply ensures that x0 < x1.
    if (x0 > x1)
    {
        float tmp = x0;
        x0 = x1;
        x1 = tmp;
    }
    return true;
}

__global__ void simplifyTimes(const float *times, const int N, const int groupSize, float *simp)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // This is done to prevent excess threads from interfering in the code.
    if (index < N)
    {
        int count = 0;
        for (int i = 0; i < groupSize; i++)
        {
            if (times[groupSize * index + i] != -1 && count < 2)
            {
                simp[2*index+count] = times[groupSize*index+i];
                count++;
            }
        }
    }
}

__global__ void forceIntersectionOrder(float *ts, Vec3<float> *coords,
                                       const int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < 2*N)
    {
        /* If the first listed intersection comes after the second
         * listed intersection, the two intersection times and
         * coordinates are swapped.
         */
        if (ts[2*index] > ts[2*index+1])
        {
            float tmpt;
            Vec3<float> tmpc;
            tmpt = ts[2*index];
            ts[2*index] = ts[2*index+1];
            ts[2*index+1] = tmpt;
            tmpc = coords[2*index];
            coords[2*index] = coords[2*index+1];
            coords[2*index+1] = tmpc;
        }
    }
}

__global__ void prepRand(hiprandState *state, int seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(((seed << 10) + idx), 0, 0, &state[idx]); 
}

__global__ void propagate(Vec3<float> *orig, float *ray_times,
                          Vec3<float> *scat_pos, float *scat_times,
                          const int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        /* Updates the neutron's main position and time
         * data with the values passed through `scat_pos`
         * and `scat_times`.
         */
        orig[index] = scat_pos[index];
        ray_times[index] = scat_times[index];
    }
}

__global__ void updateProbability(float *ray_prob,
                                  Vec3<float> *orig, Vec3<float> *int_coords,
                                  const float atten, const int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        /* Updates the neutron's probability attribute based on
         * the absorption associated with travalling through the
         * scattering body to the scattering site.
         */
        float d = (orig[index] - int_coords[2*index]).length();
        ray_prob[index] *= expf(-(d/atten));
    }
}
