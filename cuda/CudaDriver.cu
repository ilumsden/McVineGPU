#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <iomanip>
#include <limits>
#include <fstream>
//#include "H5Cpp.h"

#if defined(RANDTEST)
#include <cmath>
#endif

#include <chrono>

#include "CudaDriver.hpp"

CudaDriver::CudaDriver(std::vector< std::shared_ptr<Ray> > &rays, 
                       std::shared_ptr<AbstractShape> &shape, int bS)
{ 
    N = (int)(rays.size());
    b = shape;
    CudaErrchk( hipGetDeviceCount(&nGpu) );
    int jump = N / nGpu;
    int rem = N % nGpu;
    for (int i = 0; i <= nGpu; i++)
    {
        steps.push_back(i * jump);
    }
    for (int i = 1; i <= rem; i++)
    {
        steps[i]++;
        for (int j = i+1; j < (int)(steps.size()); j++)
        {
            steps[j]++;
        }
    }
    blockNums.resize(nGpu);
    for (int i = 0; i < (int)(steps.size())-1; i++)
    {
        blockNums[i] = ((steps[i+1] - steps[i]) + blockSize - 1) / blockSize;
    }
    // Calculates the CUDA launch parameters using bS
    /*blockSize = bS;
    numBlocks = (N + blockSize - 1) / blockSize;
    printf("CUDA Launch Params: %i, %i\n", numBlocks, blockSize);*/
    /* Allocates both host and device memory for the float arrays that
     * will be used to store the data passed to the CUDA functions.
     */
    rayptr = &rays;
    origins = (Vec3<float>*)malloc(N*sizeof(Vec3<float>));
    //CudaErrchk( hipMalloc(&d_origins, N*sizeof(Vec3<float>)) );
    vel = (Vec3<float>*)malloc(N*sizeof(Vec3<float>));
    //CudaErrchk( hipMalloc(&d_vel, N*sizeof(Vec3<float>)) );
    times = (float*)malloc(N*sizeof(float));
    //CudaErrchk( hipMalloc(&d_times, N*sizeof(float)) );
    probs = (float*)malloc(N*sizeof(float));
    //CudaErrchk( hipMalloc(&d_probs, N*sizeof(float)) );
    // Copies the data from the rays to the host arrays.
    int c = 0;
    for (auto ray : rays)
    {
        origins[c] = ray->origin;
        vel[c] = ray->vel;
        times[c] = ray->t;
        probs[c] = ray->prob;
        c++;
    }
    for (int i = 0; i < nGpu; i++)
    {
        CudaErrchk( hipSetDevice(i) );
        CudaErrchk( hipMalloc(&(d_origins[i]), (steps[i+1]-steps[i])*sizeof(Vec3<float>)) );
        CudaErrchk( hipMemcpy(d_origins[i], &(origins[steps[i]]), (steps[i+1]-steps[i])*sizeof(Vec3<float>), hipMemcpyHostToDevice) );
        CudaErrchk( hipMalloc(&(d_vel[i]), (steps[i+1]-steps[i])*sizeof(Vec3<float>)) );
        CudaErrchk( hipMemcpy(d_vel[i], &(vel[steps[i]]), (steps[i+1]-steps[i])*sizeof(Vec3<float>), hipMemcpyHostToDevice) );
        CudaErrchk( hipMalloc(&(d_times[i]), (steps[i+1]-steps[i])*sizeof(float)) );
        CudaErrchk( hipMemcpy(d_times[i], &(times[steps[i]]), (steps[i+1]-steps[i])*sizeof(float), hipMemcpyHostToDevice) );
        CudaErrchk( hipMalloc(&(d_probs[i]), (steps[i+1]-steps[i])*sizeof(float)) );
        CudaErrchk( hipMemcpy(d_probs[i], &(probs[steps[i]]), (steps[i+1]-steps[i])*sizeof(float), hipMemcpyHostToDevice) );
    }
    // Copies the data from the host arrays to the device arrays.
    //CudaErrchk( hipMemcpy(d_origins, origins, N*sizeof(Vec3<float>), hipMemcpyHostToDevice) );
    //CudaErrchk( hipMemcpy(d_vel, vel, N*sizeof(Vec3<float>), hipMemcpyHostToDevice) );
    //CudaErrchk( hipMemcpy(d_times, times, N*sizeof(float), hipMemcpyHostToDevice) );
    //CudaErrchk( hipMemcpy(d_probs, probs, N*sizeof(float), hipMemcpyHostToDevice) );
}

CudaDriver::~CudaDriver()
{
    // Frees the memory for the host-side arrays.
    free(origins);
    free(vel);
    free(times);
    free(probs);
    // Frees the memory for the device-side arrays.
    CudaErrchk( hipFree(d_origins) );
    CudaErrchk( hipFree(d_vel) );
    CudaErrchk( hipFree(d_times) );
    CudaErrchk( hipFree(d_probs) );
}

void CudaDriver::printFullData(const std::string &fname)
{
    /* If there is a file name provided (i.e. fname != str::string()),
     * the C++ stdout stream (cout) is redirected to print to the
     * desired file. Otherwise, all data is printed to stdout.
     */
    std::streambuf *coutbuf = std::cout.rdbuf();
    std::fstream fout;
    if (fname != std::string())
    {
        fout.open(fname.c_str(), std::ios::out);
        if (!fout.is_open())
        {
            std::cerr << fname << " cannot be openned.\n";
            exit(-2);
        }
        std::cout.rdbuf(fout.rdbuf());
    }
    // A generic buffer for separation purposes
    std::string buf = "        ";
    // Prints header info
    std::cout << "Position" << " " << buf << " " << buf << " || "
              << "Velocity" << " " << buf << " " << buf << " || "
              << "  Time  " << " || " << "Probability" << "\n\n";
    // Prints the data for each neutron
    for (int i = 0; i < N; i++)
    {
        std::cout << std::fixed << std::setprecision(5) << std::setw(8) << std::right
	         << origins[i][0]
		 << " " << origins[i][1]
		 << " " << origins[i][2]
		 << " || "
                 << vel[i][0]
                 << " " << vel[i][1]
                 << " " << vel[i][2]
                 << " || "
                 << times[i]
                 << " || "
                 << probs[i]
                 << "\n\n";
    }
    /* If cout was redirected, this "fixes" it so that it prints to
     * stdout in the future. Otherwise, this does nothing.
     */
    std::cout.rdbuf(coutbuf);
    // Closes the file stream if it was ever openned.
    if (fname != std::string())
    {
        fout.close();
    }
}

void CudaDriver::updateRays()
{
    for (int i = 0; i < N; i++)
    {
        (*rayptr)[i]->update(origins[i], vel[i], times[i], probs[i]);
    }
}

/*void CudaDriver::printToH5(const std::string &fname, const bool endianness)
{
    using namespace H5;
    if (fname.substr(fname.rfind(".")) != ".h5")
    {
        fprintf(stderr, "%s is not a supported HDF5 file name.\n", fname.c_str());
        return;
    }
    float data[N][4];
    for (int i = 0; i < N; i++)
    {
        data[i][0] = vel[i][0];
        data[i][1] = vel[i][1];
        data[i][2] = vel[i][2];
        data[i][3] = probs[i];
    }
    try
    {
        PredType datatype = PredType::IEEE_F32LE;
        if (endianness)
        {
            datatype = PredType::IEEE_F32BE;
        }
        Exception::dontPrint();
        H5File file(fname, H5F_ACC_TRUNC);
        hsize_t dims[2] = {N, 4};
        DataSpace dspace(2, dims);
        DataSet dset = file.createDataSet("VecProbData", datatype, dspace); 
        dset.write(data, datatype);
    }
    catch(FileIException error)
    {
        error.printErrorStack();
        //H5Eprint(stderr);
        exit(-3);
    }
    catch(DataSetIException error)
    {
        error.printErrorStack();
        //H5Eprint(stderr);
        exit(-3);
    }
    catch(DataSpaceIException error)
    {
        error.printErrorStack();
        //H5Eprint(stderr);
        exit(-3);
    }
    return;
}*/

void CudaDriver::handleExteriorIntersect(std::vector<float> &host_time,
                                         std::vector< Vec3<float> > &int_coords)
{
    /* Calls the shape's intersect function.
     * Inheritance is used to choose the correct algorithm for intersection.
     */
    b->exteriorIntersect(d_origins, d_vel, N, blockSize, numBlocks, host_time, int_coords);
    // Opens a file stream and prints the relevant data to time.txt
    // NOTE: this is for debugging purposes only. This will be removed later.
#if defined(DEBUG) || defined(PRINT1)
    std::fstream fout;
    fout.open("time.txt", std::ios::out);
    if (!fout.is_open())
    {
        std::cerr << "time.txt could not be opened.\n";
        exit(-1);
    }
    for (int i = 0; i < (int)(int_coords.size()); i++)
    {
        std::string buf = "        ";
        if (i % 2 == 0)
        {
            int ind = i/2;
            fout << "\n";
            fout << std::fixed << std::setprecision(5) << std::setw(8) << std::right
                 << origins[ind][0] << " " << origins[ind][1] << " " << origins[ind][2] << " || "
                 << vel[ind][0] << " " << vel[ind][1] << " " << vel[ind][2] << " | "
                 << host_time[i] << " / " << int_coords[i][0] << "\n";
            fout << buf << " " << buf << " " << buf << "  " << buf << " " << buf << " " << buf << " | "
                 << std::fixed << std::setprecision(5) << std::setw(8) << std::right << buf << " / " << int_coords[i][1] << "\n";
            std::string buf = "        ";
            fout << buf << " " << buf << " " << buf << "  " << buf << " " << buf << " " << buf << " | "
                 << std::fixed << std::setprecision(5) << std::setw(8) << std::right << buf << " / " << int_coords[i][2] << "\n";
        }
        else
        {
            fout << buf << " " << buf << " " << buf << "  " << buf << " " << buf << " " << buf << " | "
                 << std::fixed << std::setprecision(5) << std::setw(8) << std::right << host_time[i] << " / " << int_coords[i][0] << "\n";
            fout << buf << " " << buf << " " << buf << "  " << buf << " " << buf << " " << buf << " | "
                 << std::fixed << std::setprecision(5) << std::setw(8) << std::right << buf << " / " << int_coords[i][1] << "\n";
            fout << buf << " " << buf << " " << buf << "  " << buf << " " << buf << " " << buf << " | "
                 << std::fixed << std::setprecision(5) << std::setw(8) << std::right << buf << " / " << int_coords[i][2] << "\n";
        }
    }
    // Closes the file stream
    fout.close();
#endif
    return;
}

void CudaDriver::findScatteringSites(const std::vector<float> &int_times, 
                                     const std::vector< Vec3<float> > &int_coords)
{
#if defined(DEBUG) || defined(PRINT2)
    std::vector< Vec3<float> > tmp;
    tmp.resize(N);
    Vec3<float> *ta = tmp.data();
    memcpy(ta, origins, N*sizeof(Vec3<float>));
#endif
    // Stores the size of the `int_times` for later
    int tsize = (int)(int_times.size());
    /* Allocates memory for a device-side array that stores the
     * data passed in from `int_times`.
     */
    /*float *ts;
    CudaErrchk( hipMalloc(&ts, 2*N*sizeof(float)) );
    CudaErrchk( hipMemcpy(ts, int_times.data(), 2*N*sizeof(float), hipMemcpyHostToDevice) );*/
    std::vector<float*> ts;
    ts.resize(nGpu);
    for (int i = 0; i < nGpu; i++)
    {
        CudaErrchk( hipSetDevice(i) );
        CudaErrchk( hipMalloc(&(ts[i]), 2*(steps[i+1]-steps[i])*sizeof(float)) );
        CudaErrchk( hipMemcpy(ts[i], &(int_times.data()[steps[i]]), 2*(steps[i+1]-steps[i])*sizeof(float), hipMemcpyHostToDevice) );
    }
    /* `pos` is a device-side array that stores the coordinates of the
     * scattering sites for the neutrons.
     * The default value of its data is FLT_MAX.
     */
    /*Vec3<float> *pos;
    CudaErrchk( hipMalloc(&pos, N*sizeof(Vec3<float>)) );
    initArray< Vec3<float> ><<<numBlocks, blockSize>>>(pos, N, Vec3<float>(FLT_MAX, FLT_MAX, FLT_MAX));
    CudaErrchkNoCode();*/
    std::vector<Vec3<float>*> pos;
    pos.resize(nGpu);
    for (int i = 0; i < nGpu; i++)
    {
        CudaErrchk( hipSetDevice(i) );
        CudaErrchk( hipMalloc(&(pos[i]), (steps[i+1]-steps[i])*sizeof(Vec3<float>)) );
        initArray< Vec3<float> ><<<blockNums[i], blockSize>>>(pos[i], (steps[i+1]-steps[i]), Vec3<float>(FLT_MAX, FLT_MAX, FLT_MAX, FLT_MAX, FLT_MAX, FLT_MAX));
        CudaErrchkNoCode();
    }
    /* `scat_times` is a device_side array that stores the times at
     * which the neutrons reach their scattering sites.
     * The default value of its data is -5.
     */
    /*float *scat_times;
    CudaErrchk( hipMalloc(&scat_times, N*sizeof(float)) );
    initArray<float><<<numBlocks, blockSize>>>(scat_times, N, -5);*/
    std::vector<Vec3<float>*> scat_times;
    pos.resize(nGpu);
    for (int i = 0; i < nGpu; i++)
    {
        CudaErrchk( hipSetDevice(i) );
        CudaErrchk( hipMalloc(&(scat_times[i]), (steps[i+1]-steps[i])*sizeof(float)) );
        initArray< Vec3<float> ><<<blockNums[i], blockSize>>>(scat_times[i], (steps[i+1]-steps[i]), -5);
        CudaErrchkNoCode();
    }
    /*hiprandGenerator_t gen;
    float *d_randnums;
    CudaErrchk( hipMalloc(&d_randnums, N*sizeof(float)) );
    CuRandErrchk( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
    CuRandErrchk( hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)) );
    CuRandErrchk( hiprandGenerateUniform(gen, d_randnums, N) );*/
    std::vector<float*> d_randnums;
    d_randnums.resize(nGpu);
    for (int i = 0; i < nGpu; i++)
    {
        CudaErrchk( hipSetDevice(i) );
        CudaErrchk( hipMalloc(&(d_randnums[i]), (steps[i+1]-steps[i])*sizeof(float)) );
        hiprandGenerator_t gen;
        CuRandErrchk( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
        CuRandErrchk( hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)) );
        CuRandErrchk( hiprandGenerateUniform(gen, d_randnums[i], (steps[i+1]-steps[i])) );
        CuRandErrchk( hiprandDestroyGenerator(gen) );
    }
    // Calls the kernel for determining the scattering sites for the neutrons
    //calcScatteringSites<<<numBlocks, blockSize>>>(ts, d_origins, d_vel, pos, scat_times, d_randnums, N);
    /* Propagates the neutrons to their scattering sites.
     * In other words, the scattering coordinates and times are copied
     * into the device arrays that store the neutrons' origins and times
     * (d_origins and d_times respectively).
     */
    //propagate<<<numBlocks, blockSize>>>(d_origins, d_times, pos, scat_times, N);
    //CudaErrchkNoCode();
    /* `ic` is a device-side array that stores the intersection
     * coordinates between the neutron and scattering body, as calculated
     * in the handleIntersect function.
     */
    Vec3<float> *ic;
    CudaErrchk( hipMalloc(&ic, 2*N*sizeof(Vec3<float>)) );
    CudaErrchk( hipMemcpy(ic, int_coords.data(), 2*N*sizeof(Vec3<float>), hipMemcpyHostToDevice) );
    /* Updates the probability attribute of the neutrons to account
     * for the absorption that occurs as a neutron travels through the
     * scattering body to the scattering site.
     */
    updateProbability<<<numBlocks, blockSize>>>(d_probs, d_origins, ic, 1, 2, atten, N);
    CudaErrchkNoCode();
    // Updates the host-side arrays for the edited neutron data.
    CudaErrchk( hipMemcpy(origins, d_origins, N*sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
    CudaErrchk( hipMemcpy(times, d_times, N*sizeof(float), hipMemcpyDeviceToHost) );
    CudaErrchk( hipMemcpy(probs, d_probs, N*sizeof(float), hipMemcpyDeviceToHost) );
#if defined(DEBUG) || defined(PRINT2)
    std::fstream fout;
    fout.open("scatteringSites.txt", std::ios::out);
    if (!fout.is_open())
    {
        std::cerr << "scatteringSites.txt could not be opened.\n";
        exit(-2);
    }
    for (int i = 0; i < N; i++)
    {
        int ind = 2*i;
        fout << "\n";
        fout << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << tmp[i][0] << " " << tmp[i][1] << " " << tmp[i][2] << " || "
             << vel[i][0] << " " << vel[i][1] << " " << vel[i][2] << " || "
             << int_times[ind] << " " << int_times[ind+1] << " | "
             << origins[i][0] << "\n";
        std::string buf = "        ";
        fout << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << "   "
             << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << origins[i][1] << "\n";
        fout << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << "   "
             << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << origins[i][2] << "\n";
        fout << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << "   "
             << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << times[i] << "\n";
    }
    fout.close();
#endif
    // Frees the device memory allocated above.
    CudaErrchk( hipFree(ts) );
    CudaErrchk( hipFree(ic) );
    CudaErrchk( hipFree(pos) );
    CudaErrchk( hipFree(d_randnums) );
    return;
}

void CudaDriver::findScatteringVels()
{
#if defined(DEBUG) || defined(PRINT3)
    std::vector< Vec3<float> > tmp;
    tmp.resize(N);
    Vec3<float> *ta = tmp.data();
    memcpy(ta, vel, N*sizeof(Vec3<float>));
#endif
#if defined(DEBUG) || defined(RANDTEST)
    std::vector<float> thetas, phis;
#endif
    hiprandGenerator_t gen;
    float *d_randnums;
    CudaErrchk( hipMalloc(&d_randnums, 2*N*sizeof(float)) );
    CuRandErrchk( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
    CuRandErrchk( hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)) );
    CuRandErrchk( hiprandGenerateUniform(gen, d_randnums, 2*N) );
    /* Calls the elasticScatteringKernel function to update the neutron
     * velocities post-elastic scattering.
     */
    /*isotropicScatteringKernel<<<numBlocks, blockSize>>>(d_times,
                                                        d_vel,
                                                        d_randnums, N);*/
    scatter<<<numBlocks, blockSize>>>(0, d_times, d_vel, d_randnums, N);
    CudaErrchkNoCode();
    /* Copies the new neutron velocities into the host-side neutron
     * velocity array.
     */
    CudaErrchk( hipMemcpy(vel, d_vel, N*sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
    // Opens a file stream and prints the 
    // relevant data to scatteringVels.txt
#if defined(DEBUG) || defined(PRINT3)
    std::fstream fout;
    fout.open("scatteringVels.txt", std::ios::out);
    if (!fout.is_open())
    {
        std::cerr << "scatteringVels.txt could not be opened.\n";
        exit(-2);
    }
    for (int i = 0; i < N; i++)
    {
        fout << "\n";
        fout << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << tmp[i][0] << " " << tmp[i][1] << " " << tmp[i][2] << " || "
             << vel[i][0] << " " << vel[i][1] << " " << vel[i][2] << "\n";
    }
    fout.close();
#endif
#if defined(DEBUG) || defined(RANDTEST)
    for (int i = 0; i < N; i++)
    {
        thetas.push_back(acos(vel[i][2] / vel[i].length()));
        phis.push_back(atan2(vel[i][1], vel[i][0]));
    }
    std::sort(thetas.begin(), thetas.end());
    std::sort(phis.begin(), phis.end());
    std::fstream f1, f2;
    f1.open("thetas.txt", std::ios::out);
    if (!f1.is_open())
    {
        std::cerr << "thetas.txt could not be openned.\n";
        exit(-2);
    }
    f2.open("phis.txt", std::ios::out);
    if (!f2.is_open())
    {
        std::cerr << "phis.txt could not be openned.\n";
        exit(-2);
    }
    f1 << "Theta Values (Radians): Should range from 0 to Pi\n";
    f2 << "Phi Values (Radians): Should range from 0 to 2*Pi\n";
    for (int i = 0; i < (int)(thetas.size()); i++)
    {
        f1 << thetas[i] << "\n";
        f2 << phis[i] << "\n";
    }
    f1.close();
    f2.close();
#endif
    CuRandErrchk( hiprandDestroyGenerator(gen) );
    CudaErrchk( hipFree(d_randnums) );
}

void CudaDriver::handleInteriorIntersect()
{
#if defined(DEBUG) || defined(PRINT4)
    std::vector< Vec3<float> > tmp;
    tmp.resize(N);
    Vec3<float> *ta = tmp.data();
    memcpy(ta, origins, N*sizeof(Vec3<float>));
#endif
    std::vector<float> int_times;
    std::vector< Vec3<float> > int_coords;
    b->interiorIntersect(d_origins, d_vel, N, blockSize, numBlocks, int_times, int_coords); 
    float *exit_times;
    CudaErrchk( hipMalloc(&exit_times, N*sizeof(float)) );
    CudaErrchk( hipMemcpy(exit_times, int_times.data(), N*sizeof(float), hipMemcpyHostToDevice) );
    Vec3<float> *exit_coords;
    CudaErrchk( hipMalloc(&exit_coords, N*sizeof(Vec3<float>)) );
    CudaErrchk( hipMemcpy(exit_coords, int_coords.data(), N*sizeof(Vec3<float>), hipMemcpyHostToDevice) );
    updateProbability<<<numBlocks, blockSize>>>(d_probs, exit_coords, d_origins, 1, 1, atten, N);
    propagate<<<numBlocks, blockSize>>>(d_origins, d_times, exit_coords, exit_times, N);
    CudaErrchk( hipMemcpy(origins, d_origins, N*sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
    CudaErrchk( hipMemcpy(times, d_times, N*sizeof(float), hipMemcpyDeviceToHost) );
    CudaErrchk( hipMemcpy(probs, d_probs, N*sizeof(float), hipMemcpyDeviceToHost) );
#if defined(DEBUG) || defined(PRINT4)
    std::fstream fout;
    fout.open("exit.txt", std::ios::out);
    if (!fout.is_open())
    {
        std::cerr << "exit.txt could not be openned.\n";
        exit(-2);
    }
    for (int i = 0; i < N; i++)
    {
        std::string buf = "        ";
        fout << "\n";
        fout << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << tmp[i][0] << " " << tmp[i][1] << " " << tmp[i][2] << " || "
             << vel[i][0] << " " << vel[i][1] << " " << vel[i][2] << " | "
             << origins[i][0] << "\n";
        fout << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << " " << buf << "   "
             << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << origins[i][1] << "\n";
        fout << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << " " << buf << "   "
             << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << origins[i][2] << "\n";
        fout << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << " " << buf << "   "
             << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << times[i] << "\n";
        fout << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << " " << buf << "   "
             << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << probs[i] << "\n";
    }
    fout.close();
#endif
    CudaErrchk( hipFree(exit_times) );
    CudaErrchk( hipFree(exit_coords) );
}

void CudaDriver::runCalculations()
{
    /* Creates the vectors that will store the intersection
     * times and coordinates.
     */
    std::vector<float> int_times;
    std::vector< Vec3<float> > int_coords;
    // Starts the intersection calculation
    handleExteriorIntersect(int_times, int_coords);
    // Starts the scattering site calculation
    findScatteringSites(int_times, int_coords);
    // Starts the elastic scattering calculation
    findScatteringVels();
    handleInteriorIntersect();
}

std::ostream& operator<<(std::ostream &fout, const CudaDriver &cd)
{
    std::vector<float> data;
    for (int i = 0; i < cd.N; i++)
    {
        data.push_back(cd.vel[i][0]);
        data.push_back(cd.vel[i][1]);
        data.push_back(cd.vel[i][2]);
        data.push_back(cd.probs[i]);
    }
    char *bytes = (char*)(data.data());
    fout.write(bytes, ((int)(data.size()))*sizeof(float));
    return fout;
}
