#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <iomanip>
#include <fstream>

#if defined(RANDTEST)
#include <cmath>
#endif

#include <chrono>

#include "CudaDriver.hpp"

CudaDriver::CudaDriver(std::vector< std::shared_ptr<Ray> > &rays, 
                       std::shared_ptr<AbstractShape> &shape, int bS)
{ 
    N = (int)(rays.size());
    b = shape;
    // Calculates the CUDA launch parameters using bS
    blockSize = bS;
    numBlocks = (N + blockSize - 1) / blockSize;
    printf("blockSize = %i\nnumBlocks = %i\n", blockSize, numBlocks);
    /* Allocates both host and device memory for the float arrays that
     * will be used to store the data passed to the CUDA functions.
     */
    rayptr = &rays;
    origins = (Vec3<float>*)malloc(N*sizeof(Vec3<float>));
    CudaErrchk( hipMalloc(&d_origins, N*sizeof(Vec3<float>)) );
    vel = (Vec3<float>*)malloc(N*sizeof(Vec3<float>));
    CudaErrchk( hipMalloc(&d_vel, N*sizeof(Vec3<float>)) );
    times = (float*)malloc(N*sizeof(float));
    CudaErrchk( hipMalloc(&d_times, N*sizeof(float)) );
    probs = (float*)malloc(N*sizeof(float));
    CudaErrchk( hipMalloc(&d_probs, N*sizeof(float)) );
    // Copies the data from the rays to the host arrays.
    int c = 0;
    for (auto ray : rays)
    {
        origins[c] = ray->origin;
        vel[c] = ray->vel;
        times[c] = ray->t;
        probs[c] = ray->prob;
        c++;
    }
    // Copies the data from the host arrays to the device arrays.
    CudaErrchk( hipMemcpy(d_origins, origins, N*sizeof(Vec3<float>), hipMemcpyHostToDevice) );
    CudaErrchk( hipMemcpy(d_vel, vel, N*sizeof(Vec3<float>), hipMemcpyHostToDevice) );
    CudaErrchk( hipMemcpy(d_times, times, N*sizeof(float), hipMemcpyHostToDevice) );
    CudaErrchk( hipMemcpy(d_probs, probs, N*sizeof(float), hipMemcpyHostToDevice) );
}

CudaDriver::~CudaDriver()
{
    // Frees the memory for the host-side arrays.
    free(origins);
    free(vel);
    free(times);
    free(probs);
    // Frees the memory for the device-side arrays.
    CudaErrchk( hipFree(d_origins) );
    CudaErrchk( hipFree(d_vel) );
    CudaErrchk( hipFree(d_times) );
    CudaErrchk( hipFree(d_probs) );
}

void CudaDriver::printData(const std::string &fname)
{
    /* If there is a file name provided (i.e. fname != str::string()),
     * the C++ stdout stream (cout) is redirected to print to the
     * desired file. Otherwise, all data is printed to stdout.
     */
    std::streambuf *coutbuf = std::cout.rdbuf();
    std::fstream fout;
    if (fname != std::string())
    {
        fout.open(fname.c_str(), std::ios::out);
        if (!fout.is_open())
        {
            std::cerr << fname << " cannot be openned.\n";
            exit(-2);
        }
        std::cout.rdbuf(fout.rdbuf());
    }
    // A generic buffer for separation purposes
    std::string buf = "        ";
    // Prints header info
    std::cout << "Position" << " " << buf << " " << buf << " || "
              << "Velocity" << " " << buf << " " << buf << " || "
              << "  Time  " << " || " << "Probability" << "\n\n";
    // Prints the data for each neutron
    for (int i = 0; i < N; i++)
    {
        std::cout << std::fixed << std::setprecision(5) << std::setw(8) << std::right
	         << origins[i][0]
		 << " " << origins[i][1]
		 << " " << origins[i][2]
		 << " || "
                 << vel[i][0]
                 << " " << vel[i][1]
                 << " " << vel[i][2]
                 << " || "
                 << times[i]
                 << " || "
                 << probs[i]
                 << "\n\n";
    }
    /* If cout was redirected, this "fixes" it so that it prints to
     * stdout in the future. Otherwise, this does nothing.
     */
    std::cout.rdbuf(coutbuf);
    // Closes the file stream if it was ever openned.
    if (fname != std::string())
    {
        fout.close();
    }
}

void CudaDriver::handleExteriorIntersect(//std::shared_ptr<AbstractShape> &b, 
                                         std::vector<float> &host_time,
                                         std::vector< Vec3<float> > &int_coords)
{
    /* Calls the shape's intersect function.
     * Inheritance is used to choose the correct algorithm for intersection.
     */
    b->exteriorIntersect(d_origins, d_vel, N, blockSize, numBlocks, host_time, int_coords);
    // Opens a file stream and prints the relevant data to time.txt
    // NOTE: this is for debugging purposes only. This will be removed later.
#if defined(DEBUG) || defined(PRINT1)
    std::fstream fout;
    fout.open("time.txt", std::ios::out);
    if (!fout.is_open())
    {
        std::cerr << "time.txt could not be opened.\n";
        exit(-1);
    }
    for (int i = 0; i < (int)(int_coords.size()); i++)
    {
        std::string buf = "        ";
        if (i % 2 == 0)
        {
            int ind = i/2;
            fout << "\n";
            fout << std::fixed << std::setprecision(5) << std::setw(8) << std::right
                 << origins[ind][0] << " " << origins[ind][1] << " " << origins[ind][2] << " || "
                 << vel[ind][0] << " " << vel[ind][1] << " " << vel[ind][2] << " | "
                 << host_time[i] << " / " << int_coords[i][0] << "\n";
            fout << buf << " " << buf << " " << buf << "  " << buf << " " << buf << " " << buf << " | "
                 << std::fixed << std::setprecision(5) << std::setw(8) << std::right << buf << " / " << int_coords[i][1] << "\n";
            std::string buf = "        ";
            fout << buf << " " << buf << " " << buf << "  " << buf << " " << buf << " " << buf << " | "
                 << std::fixed << std::setprecision(5) << std::setw(8) << std::right << buf << " / " << int_coords[i][2] << "\n";
        }
        else
        {
            fout << buf << " " << buf << " " << buf << "  " << buf << " " << buf << " " << buf << " | "
                 << std::fixed << std::setprecision(5) << std::setw(8) << std::right << host_time[i] << " / " << int_coords[i][0] << "\n";
            fout << buf << " " << buf << " " << buf << "  " << buf << " " << buf << " " << buf << " | "
                 << std::fixed << std::setprecision(5) << std::setw(8) << std::right << buf << " / " << int_coords[i][1] << "\n";
            fout << buf << " " << buf << " " << buf << "  " << buf << " " << buf << " " << buf << " | "
                 << std::fixed << std::setprecision(5) << std::setw(8) << std::right << buf << " / " << int_coords[i][2] << "\n";
        }
    }
    // Closes the file stream
    fout.close();
#endif
    return;
}

void CudaDriver::findScatteringSites(const std::vector<float> &int_times, 
                                     const std::vector< Vec3<float> > &int_coords)
{
#if defined(DEBUG) || defined(PRINT2)
    std::vector< Vec3<float> > tmp;
    tmp.resize(N);
    Vec3<float> *ta = tmp.data();
    memcpy(ta, origins, N*sizeof(Vec3<float>));
#endif
    // Stores the size of the `int_times` for later
    int tsize = (int)(int_times.size());
    /* Allocates memory for a device-side array that stores the
     * data passed in from `int_times`.
     */
    float *ts;
    CudaErrchk( hipMalloc(&ts, 2*N*sizeof(float)) );
    CudaErrchk( hipMemcpy(ts, int_times.data(), 2*N*sizeof(float), hipMemcpyHostToDevice) );
    /* `pos` is a device-side array that stores the coordinates of the
     * scattering sites for the neutrons.
     * The default value of its data is FLT_MAX.
     */
    Vec3<float> *pos;
    CudaErrchk( hipMalloc(&pos, N*sizeof(Vec3<float>)) );
    initArray< Vec3<float> ><<<numBlocks, blockSize>>>(pos, N, Vec3<float>(FLT_MAX, FLT_MAX, FLT_MAX));
    CudaErrchkNoCode();
    /* `scat_times` is a device_side array that stores the times at
     * which the neutrons reach their scattering sites.
     * The default value of its data is -5.
     */
    float *scat_times;
    CudaErrchk( hipMalloc(&scat_times, N*sizeof(float)) );
    initArray<float><<<numBlocks, blockSize>>>(scat_times, N, -5);
    hiprandState *state;
    CudaErrchk( hipMalloc(&state, numBlocks*blockSize*sizeof(hiprandState)) );
    auto start = std::chrono::steady_clock::now();
    prepRand<<<numBlocks, blockSize>>>(state, time(NULL));
    CudaErrchkNoCode();
    auto stop = std::chrono::steady_clock::now();
    double time = std::chrono::duration<double>(stop - start).count();
    printf("Rand Prep Complete\n    Summary: Time = %f\n", time);
    // Calls the kernel for determining the scattering sites for the neutrons
    calcScatteringSites<<<numBlocks, blockSize>>>(ts, d_origins, d_vel, pos, scat_times, state, N);
    /* Propagates the neutrons to their scattering sites.
     * In other words, the scattering coordinates and times are copied
     * into the device arrays that store the neutrons' origins and times
     * (d_origins and d_times respectively).
     */
    propagate<<<numBlocks, blockSize>>>(d_origins, d_times, pos, scat_times, N);
    CudaErrchkNoCode();
    /* `ic` is a device-side array that stores the intersection
     * coordinates between the neutron and scattering body, as calculated
     * in the handleIntersect function.
     */
    Vec3<float> *ic;
    CudaErrchk( hipMalloc(&ic, 2*N*sizeof(Vec3<float>)) );
    CudaErrchk( hipMemcpy(ic, int_coords.data(), 2*N*sizeof(Vec3<float>), hipMemcpyHostToDevice) );
    /* Updates the probability attribute of the neutrons to account
     * for the absorption that occurs as a neutron travels through the
     * scattering body to the scattering site.
     */
    updateProbability<<<numBlocks, blockSize>>>(d_probs, d_origins, ic, 1, 2, atten, N);
    CudaErrchkNoCode();
    // Updates the host-side arrays for the edited neutron data.
    CudaErrchk( hipMemcpy(origins, d_origins, N*sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
    CudaErrchk( hipMemcpy(times, d_times, N*sizeof(float), hipMemcpyDeviceToHost) );
    CudaErrchk( hipMemcpy(probs, d_probs, N*sizeof(float), hipMemcpyDeviceToHost) );
#if defined(DEBUG) || defined(PRINT2)
    std::fstream fout;
    fout.open("scatteringSites.txt", std::ios::out);
    if (!fout.is_open())
    {
        std::cerr << "scatteringSites.txt could not be opened.\n";
        exit(-2);
    }
    for (int i = 0; i < N; i++)
    {
        int ind = 2*i;
        fout << "\n";
        fout << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << tmp[i][0] << " " << tmp[i][1] << " " << tmp[i][2] << " || "
             << vel[i][0] << " " << vel[i][1] << " " << vel[i][2] << " || "
             << int_times[ind] << " " << int_times[ind+1] << " | "
             << origins[i][0] << "\n";
        std::string buf = "        ";
        fout << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << "   "
             << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << origins[i][1] << "\n";
        fout << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << "   "
             << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << origins[i][2] << "\n";
        fout << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << "   "
             << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << times[i] << "\n";
    }
    fout.close();
#endif
    // Frees the device memory allocated above.
    CudaErrchk( hipFree(ts) );
    CudaErrchk( hipFree(ic) );
    CudaErrchk( hipFree(pos) );
    CudaErrchk( hipFree(state) );
    return;
}

void CudaDriver::findScatteringVels()
{
#if defined(DEBUG) || defined(PRINT3)
    std::vector< Vec3<float> > tmp;
    tmp.resize(N);
    Vec3<float> *ta = tmp.data();
    memcpy(ta, vel, N*sizeof(Vec3<float>));
#endif
#if defined(DEBUG) || defined(RANDTEST)
    std::vector<float> thetas, phis;
#endif
    hiprandState *state;
    CudaErrchk( hipMalloc(&state, numBlocks*blockSize*sizeof(hiprandState)) );
    auto start = std::chrono::steady_clock::now();
    prepRand<<<numBlocks, blockSize>>>(state, time(NULL));
    CudaErrchkNoCode();
    auto stop = std::chrono::steady_clock::now();
    double time = std::chrono::duration<double>(stop - start).count();
    printf("Rand Prep Complete\n    Summary: Time = %f\n", time);
    /* Calls the elasticScatteringKernel function to update the neutron
     * velocities post-elastic scattering.
     */
    elasticScatteringKernel<<<numBlocks, blockSize>>>(d_times,
                                                      d_vel,
                                                      state, N);
    CudaErrchkNoCode();
    /* Copies the new neutron velocities into the host-side neutron
     * velocity array.
     */
    CudaErrchk( hipMemcpy(vel, d_vel, N*sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
    // Opens a file stream and prints the 
    // relevant data to scatteringVels.txt
#if defined(DEBUG) || defined(PRINT3)
    std::fstream fout;
    fout.open("scatteringVels.txt", std::ios::out);
    if (!fout.is_open())
    {
        std::cerr << "scatteringVels.txt could not be opened.\n";
        exit(-2);
    }
    for (int i = 0; i < N; i++)
    {
        fout << "\n";
        fout << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << tmp[i][0] << " " << tmp[i][1] << " " << tmp[i][2] << " || "
             << vel[i][0] << " " << vel[i][1] << " " << vel[i][2] << "\n";
    }
    fout.close();
#endif
#if defined(DEBUG) || defined(RANDTEST)
    for (int i = 0; i < N; i++)
    {
        thetas.push_back(acos(vel[i][2] / vel[i].length()));
        phis.push_back(atan2(vel[i][1], vel[i][0]));
    }
    std::sort(thetas.begin(), thetas.end());
    std::sort(phis.begin(), phis.end());
    std::fstream f1, f2;
    f1.open("thetas.txt", std::ios::out);
    if (!f1.is_open())
    {
        std::cerr << "thetas.txt could not be openned.\n";
        exit(-2);
    }
    f2.open("phis.txt", std::ios::out);
    if (!f2.is_open())
    {
        std::cerr << "phis.txt could not be openned.\n";
        exit(-2);
    }
    f1 << "Theta Values (Radians): Should range from 0 to Pi\n";
    f2 << "Phi Values (Radians): Should range from 0 to 2*Pi\n";
    for (int i = 0; i < (int)(thetas.size()); i++)
    {
        f1 << thetas[i] << "\n";
        f2 << phis[i] << "\n";
    }
    f1.close();
    f2.close();
#endif
    CudaErrchk( hipFree(state) );
}

void CudaDriver::handleInteriorIntersect()
{
#if defined(DEBUG) || defined(PRINT4)
    std::vector< Vec3<float> > tmp;
    tmp.resize(N);
    Vec3<float> *ta = tmp.data();
    memcpy(ta, origins, N*sizeof(Vec3<float>));
#endif
    std::vector<float> int_times;
    std::vector< Vec3<float> > int_coords;
    b->interiorIntersect(d_origins, d_vel, N, blockSize, numBlocks, int_times, int_coords); 
    float *exit_times;
    CudaErrchk( hipMalloc(&exit_times, N*sizeof(float)) );
    CudaErrchk( hipMemcpy(exit_times, int_times.data(), N*sizeof(float), hipMemcpyHostToDevice) );
    Vec3<float> *exit_coords;
    CudaErrchk( hipMalloc(&exit_coords, N*sizeof(Vec3<float>)) );
    CudaErrchk( hipMemcpy(exit_coords, int_coords.data(), N*sizeof(Vec3<float>), hipMemcpyHostToDevice) );
    updateProbability<<<numBlocks, blockSize>>>(d_probs, exit_coords, d_origins, 1, 1, atten, N);
    propagate<<<numBlocks, blockSize>>>(d_origins, d_times, exit_coords, exit_times, N);
    CudaErrchk( hipMemcpy(origins, d_origins, N*sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
    CudaErrchk( hipMemcpy(times, d_times, N*sizeof(float), hipMemcpyDeviceToHost) );
    CudaErrchk( hipMemcpy(probs, d_probs, N*sizeof(float), hipMemcpyDeviceToHost) );
#if defined(DEBUG) || defined(PRINT4)
    std::fstream fout;
    fout.open("exit.txt", std::ios::out);
    if (!fout.is_open())
    {
        std::cerr << "exit.txt could not be openned.\n";
        exit(-2);
    }
    for (int i = 0; i < N; i++)
    {
        std::string buf = "        ";
        fout << "\n";
        fout << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << tmp[i][0] << " " << tmp[i][1] << " " << tmp[i][2] << " || "
             << vel[i][0] << " " << vel[i][1] << " " << vel[i][2] << " | "
             << origins[i][0] << "\n";
        fout << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << " " << buf << "   "
             << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << origins[i][1] << "\n";
        fout << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << " " << buf << "   "
             << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << origins[i][2] << "\n";
        fout << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << " " << buf << "   "
             << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << times[i] << "\n";
        fout << buf << " " << buf << " " << buf << "    "
             << buf << " " << buf << " " << buf << "   "
             << std::fixed << std::setprecision(5) << std::setw(8) << std::right
             << probs[i] << "\n";
    }
    fout.close();
#endif
    CudaErrchk( hipFree(exit_times) );
    CudaErrchk( hipFree(exit_coords) );
}

void CudaDriver::runCalculations()
{
    /* Creates the vectors that will store the intersection
     * times and coordinates.
     */
    std::vector<float> int_times;
    std::vector< Vec3<float> > int_coords;
    // Starts the intersection calculation
    auto start = std::chrono::steady_clock::now();
    handleExteriorIntersect(int_times, int_coords);
    auto stop = std::chrono::steady_clock::now();
    double time = std::chrono::duration<double>(stop - start).count();
    printf("handleExteriorIntersect: %f\n", time);
    // Starts the scattering site calculation
    start = std::chrono::steady_clock::now();
    findScatteringSites(int_times, int_coords);
    stop = std::chrono::steady_clock::now();
    time = std::chrono::duration<double>(stop - start).count();
    printf("findScatteringSites: %f\n", time);
    // Starts the elastic scattering calculation
    start = std::chrono::steady_clock::now();
    findScatteringVels();
    stop = std::chrono::steady_clock::now();
    time = std::chrono::duration<double>(stop - start).count();
    printf("findScatteringVels: %f\n", time);
    start = std::chrono::steady_clock::now();
    handleInteriorIntersect();
    stop = std::chrono::steady_clock::now();
    time = std::chrono::duration<double>(stop - start).count();
    printf("handleInteriorIntersect: %f\n", time);
}
