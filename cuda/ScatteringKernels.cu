#include "hip/hip_runtime.h"
#include "ScatteringKernels.hpp"

__device__ void randCoord(Vec3<float> *inters, float *time,
                          Vec3<float> &pos,
                          hiprandState *state)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    /* Instead of pasing the initial ray data, the two intersection
     * points and times are used to recalculate the velocities.
     */
    float dt = time[1] - time[0];
    Vec3<float> m = (inters[1] - inters[0]) * (1.0/dt);
    // cuRand is used to generate a random time between 0 and dt.
    float randt = hiprand_uniform(&(state[index]));
    randt *= dt;
    /* Basic kinematics are used to calculate the coordinates of
     * the randomly chosen scattering site.
     */
    pos = inters[0] + (m*randt);
}

__global__ void calcScatteringSites(float *ts, Vec3<float> *int_pts,
                                    Vec3<float> *pos, hiprandState *state,
                                    const int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // This is done to prevent excess threads from interfering in the code.
    if (index < N)
    {
        /* If the intersection times for the neutron are the default
         * value of -5, there was no intersection, so the function
         * terminates.
         */
        if (ts[2*index] != -5 && ts[2*index+1] != -5)
        {
            /* The randCoord function assumes that the first time
             * is smaller than the second. If this is not the
             * case, the times and the corresponding intersection
             * coordinates are swapped.
             */
            if (ts[2*index] > ts[2*index+1])
            {
                float tmpt;
                Vec3<float> tmpv;
                tmpt = ts[2*index];
                ts[2*index] = ts[2*index+1];
                ts[2*index+1] = tmpt;
                tmpv = int_pts[2*index];
                int_pts[2*index] = int_pts[2*index+1];
                int_pts[2*index+1] = tmpv;
            }
            /* The randCoord function is called to determine the
             * scattering site.
             */
            randCoord(&(int_pts[2*index]), &(ts[2*index]), pos[index], state);
        }
    }
}

__global__ void elasticScatteringKernel(const Vec3<float> *initVel,
                                        Vec3<float> *postVel,
                                        hiprandState *state,
                                        const int N)
{
    /* To start each hiprandState will be used to generate the random
     * z and phi values.
     */
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        float z = hiprand_uniform(state[index]);
        z *= 2;
        z -= 1;
        float phi = hiprand_uniform(state[index]);
        phi *= 2*PI;
        float theta = acosf(z);
        float rho = sqrtf(initVel[0]*initVel[0] +
                          initVel[1]*initVel[1] +
                          initVel[2]*initVel[2]);
        postVel[index][0] = rho * sinf(phi) * cosf(theta);
        postVel[index][1] = rho * sinf(phi) * sinf(theta);
        postVel[index][2] = rho * z;
    }
}
