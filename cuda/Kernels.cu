#include "hip/hip_runtime.h"
#include <cstdio>

#include "Kernels.hpp"

__global__ void initArray(float *data, int size, const float val)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < size; i += stride)
    {
        data[i] = val;
    }
}

__device__ float dot(float ax, float ay, float az,
                     float bx, float by, float bz)
{
    return ax*bx + ay*by + az*bz;
}

__device__ void cross(float ax, float ay, float az,
                      float bx, float by, float bz,
                      float *cx, float *cy, float *cz)
{
    *cx = ay*bz - az*by;
    *cy = az*bx - ax*bz;
    *cz = ax*by - by*bx;
    return;
}

/* This is a device-only helper function for determining the time
 * it takes a ray to intersect the rectangle specified by the `intersectRectangle`
 * function.
 * It is a CUDA version of the intersectRectangle function from ArrowIntersector.cc
 * in McVine (mcvine/packages/mccomposite/lib/geometry/visitors/ArrowIntersector.cc).
 */
__device__ void intersectRectangle(float* ts, float* pts,
                                   float x, float y, float z, float zdiff,
                                   float va, float vb, float vc, 
                                   const float A, const float B,
                                   const int key, const int groupSize, 
                                   const int off1, int &off2)
{
    z -= zdiff;
    float t = (0-z)/vc;
    float r1x = x+va*t; 
    float r1y = y+vb*t;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (fabsf(r1x) < (A/2) && fabsf(r1y) < (B/2))
    {
        float ix, iy, iz;
        if (key == 0)
        {
            ix = r1x;
            iy = r1y;
            iz = zdiff;
        }
        else if (key == 1)
        {
            iy = r1x;
            iz = r1y;
            ix = zdiff;
        }
        else
        {
            iz = r1x;
            ix = r1y;
            iy = zdiff;
        }
        if (off2 == 0 || off2 == 3)
        {
            pts[6*index + off2] = ix;
            pts[6*index + off2 + 1] = iy;
            pts[6*index + off2 + 2] = iz;
            off2 += 3;
            //printf("Rectangle: index = %i    off2 = %i\n", index, off2);
        }
        //ascii(r) = 114
        ts[off1 + index*groupSize] = t + 114;
    }
    else
    {
        ts[off1 + index*groupSize] = -1;
    }
}

__device__ void intersectCylinderSide(float *ts, float *pts,
                                      float x, float y, float z,
                                      float vx, float vy, float vz,
                                      const float r, const float h, 
                                      int &offset)
{
    float a = vx*vx + vy*vy;
    float b = x*vx + y*vy;
    float c = x*x+y*y - r*r;
    float k = b*b - a*c;
    float t;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (k < 0)
    {
        ts[4*index + 2] = -1;
        ts[4*index + 3] = -1;
        return;
    }
    else if (k == 0)
    {
        t = -b/a;
        ts[4*index + 3] = -1;
        if (fabsf(z+vz*t) < h/2)
        {
            ts[4*index + 2] = t;
            if (offset == 0 || offset == 3)
            {
                pts[6*index + offset] = x+vx*t;
                pts[6*index + offset + 1] = y+vy*t;
                pts[6*index + offset + 2] = z+vz*t;
                offset += 3;
            }
        }
        else
        {
            ts[4*index + 2] = -1;
        }
    }
    __syncthreads();
    int i = 2;
    k = sqrtf(k);
    t = (-b+k)/a;
    if (fabsf(z+vz*t) < h/2)
    {
        ts[4*index + i] = t;
        i++;
        if (offset == 0 || offset == 3)
        {
            pts[6*index + offset] = x+vx*t;
            pts[6*index + offset + 1] = y+vy*t;
            pts[6*index + offset + 2] = z+vz*t;
            offset += 3;
        }
    }
    t = (-b-k)/a;
    if (fabsf(z+vz*t) < h/2)
    {
        ts[4*index + i] = t;
        i++;
        if (offset == 0 || offset == 3)
        {
            pts[6*index + offset] = x+vx*t;
            pts[6*index + offset + 1] = y+vy*t;
            pts[6*index + offset + 2] = z+vz*t;
            offset += 3;
        }
    }
    if (i < 4)
    {
        for (int j = i; j < 4; j++)
        {
            ts[4*index + j] = -1;
        }
    }
    __syncthreads();
}

__device__ void intersectCylinderTopBottom(float *ts, float *pts,
                                           float x, float y, float z,
                                           float vx, float vy, float vz,
                                           const float r, const float h,
                                           int &offset)
{
    float r2 = r*r;
    float hh = h/2;
    float x1, y1;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float t = (hh-z)/vz;
    x1 = x + vx*t;
    y1 = y + vy*t;
    if (x1*x1 + y1*y1 <= r2)
    {
        ts[4*index] = t;
        if (offset == 0 || offset == 3)
        {
            pts[6*index + offset] = x1;
            pts[6*index + offset + 1] = y1;
            pts[6*index + offset + 2] = hh;
            offset += 3;
        }
    }
    else
    {
        ts[4*index] = -1;
    }
    t = (-hh-z)/vz;
    x1 = x + vx*t;
    y1 = y + vy*t;
    if (x1*x1 + y1*y1 <= r2)
    {
        ts[4*index + 1] = t;
        if (offset == 0 || offset == 3)
        {
            pts[6*index + offset] = x1;
            pts[6*index + offset + 1] = y1;
            pts[6*index + offset + 2] = -hh;
            offset += 3;
        }
    }
    else
    {
        ts[4*index + 1] = -1;
    }
}

__device__ void intersectTriangle(float *ts, float *pts,
                                  const float x, const float y, const float z,
                                  const float vx, const float vy, const float vz,
                                  const float aX, const float aY, const float aZ, 
                                  const float bX, const float bY, const float bZ,
                                  const float cX, const float cY, const float cZ,
                                  const int off1, int &off2)
{   
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float abX = bX - aX, abY = bY - aY, abZ = bZ - aZ;
    float acX = cX - aX, acY = cY - aY, acZ = cZ - aZ;
    float nX, nY, nZ;
    cross(abX, abY, abZ, acX, acY, acZ, &nX, &nY, &nZ);
    float nLength = fabsf(nX)*fabsf(nX)+fabsf(nY)*fabsf(nY)+fabsf(nZ)*fabsf(nZ);
    nLength = sqrtf(nLength);
    nX /= nLength; nY /= nLength; nZ /= nLength;
    float ndv = dot(nX, nY, nZ, vx, vy, vz);
    if (fabsf(ndv) < 1e-10)
    {
        ts[5*index + off1] = -1;
        return;
    }
    float d = dot(nX, nY, nZ, aX, aY, aZ);
    float t = -(dot(nX, nY, nZ, x, y, z) + d) / ndv;
    if (t < 0)
    {
        ts[5*index + off1] = -1;
        return;
    }
    float pX = x + vx*t, pY = y + vy*t, pZ = z + vz*t;
    float apX = pX - aX, apY = pY - aY, apZ = pZ - aZ;
    float edge1X = cX - bX, edge1Y = cY - bY, edge1Z = cZ - bZ;
    float bpX = pX - bX, bpY = pY - bY, bpZ = pZ - bZ;
    float cpX = pX - cX, cpY = pY - cY, cpZ = pZ - cZ;
    float c0X, c0Y, c0Z, c1X, c1Y, c1Z, c2X, c2Y, c2Z;
    cross(abX, abY, abZ, apX, apY, apZ, &c0X, &c0Y, &c0Z);
    cross(edge1X, edge1Y, edge1Z, bpX, bpY, bpZ, &c1X, &c1Y, &c1Z);
    cross(-acX, -acY, -acZ, cpX, cpY, cpZ, &c2X, &c2Y, &c2Z);
    if (dot(nX, nY, nZ, c0X, c0Y, c0Z) < 0 ||
        dot(nX, nY, nZ, c1X, c1Y, c1Z) < 0 ||
        dot(nX, nY, nZ, c2X, c2Y, c2Z) < 0)
    {
        ts[5*index+off1] = -1;
        return;
    }
    ts[5*index + off1] = t;
    if (off2 == 0 || off2 == 3)
    {
        pts[6*index + off2] = pX;
        pts[6*index + off2 + 1] = pY;
        pts[6*index + off2 + 2] = pZ;
        off2 += 3;
    }
    __syncthreads();
    return;
    /*int index = blockIdx.x * blockDim.x + threadIdx.x;
    float abX = bX - aX, abY = bY - aY, abZ = bZ - aZ;
    float acX = cX - aX, acY = cY - aY, acZ = cZ - aZ;
    float nX, nY, nZ;
    cross(abX, abY, abZ, acX, acY, acZ, &nX, &nY, &nZ);
    float nLength = fabsf(nX)*fabsf(nX)+fabsf(nY)*fabsf(nY)+fabsf(nZ)*fabsf(nZ);
    nLength = sqrtf(nLength);
    nX /= nLength; nY /= nLength; nZ /= nLength;
    float d = dot(nX, nY, nZ, aX, aY, aZ);
    float v_p = dot(nX, nY, nZ, vx, vy, vz);
    if (fabsf(v_p) < 1e-10)
    {
        ts[5*index + off1] = -1;
        return;
    }
    float r_p = dot(nX, nY, nZ, x, y, z);
    float t = (d - r_p)/v_p;
    //printf("index = %i\n    abX = %f abY = %f abZ = %f\n    acX = %f acY = %f acZ = %f\n    nX = %f nY = %f nZ = %f\n    d = %f r_p = %f v_p = %f\n    t = %f\n", index, abX, abY, abZ, acX, acY, acZ, nX, nY, nZ, d, r_p, v_p, t);
    float pX = x + vx*t, pY = y + vy*t, pZ = z + vz*t;
    float apX = pX - aX, apY = pY - aY, apZ = pZ - aZ;
    float ncX, ncY, ncZ;
    cross(nX, nY, nZ, acX, acY, acZ, &ncX, &ncY, &ncZ);
    float c1 = dot(apX, apY, apZ, ncX, ncY, ncZ)/dot(abX, abY, abZ, ncX, ncY, ncZ);
    if (c1 < 0)
    {
        ts[5*index + off1] = -1;
        return;
    }
    float nbX, nbY, nbZ;
    cross(nX, nY, nZ, abX, abY, abZ, &nbX, &nbY, &nbZ);
    float c2 = dot(apX, apY, apZ, nbX, nbY, nbZ)/dot(acX, acY, acZ, nbX, nbY, nbZ);
    if (c2 < 0)
    {
        ts[5*index + off1] = -1;
        return;
    }
    if (c1+c2 > 1)
    {
        ts[5*index + off1] = -1;
        return;
    }
    // Set time to actual value and record pX, pY, and pZ as int pts.
    // ascii(T) = 84
    ts[5*index + off1] = t + 84;
    if (off2 == 0 || off2 == 3)
    {
        pts[6*index + off2] = pX;
        pts[6*index + off2 + 1] = pY;
        pts[6*index + off2 + 2] = pZ;
        //printf("index = %i: time = %f\n    x = %f y = %f z = %f\n    vx = %f vy = %f vz = %f\n    pX = %f pY = %f pZ = %f\n    pts[%i] = %f pts[%i] = %f pts[%i] = %f\n", index, t, x, y, z, vx, vy, vz, pX, pY, pZ, 6*index+off2, pts[6*index + off2], 6*index+off2+1, pts[6*index + off2+1], 6*index+off2+2, pts[6*index + off2+2]);
        off2 += 3;
        //printf("Triangle: index = %i    off2 = %i\n", index, off2);
    }
    __syncthreads();*/
    return;
}

/*__device__ void calculateQuadCoef(float x, float vx, float vy, float vz,
                                  float dist, float &disc,
                                  float &a, float &b, float &c)
{
    a = 1 + (vy/vx)*(vy/vx) + (vz/vx)*(vz/vx);
    b = -2*(1 + ((x*vy*vy)/(vx*vx)) + ((x*vz*vz)/(vx*vx)));
    c = x*x + ((x*vy)/vx)*((x*vy)/vx) + ((x*vz)/vx)*((x*vz)/vx);
    c -= dist*dist;
    disc = b*b - 4*a*c;
    return;
}*/

__device__ bool solveQuadratic(float a, float b, float c, float &x0, float &x1)
{
    float discr = b*b - 4*a*c;
    if (discr < 0)
    {
        return false;
    }
    else
    {
        // Done to avoid "catastrophic cancellation"
        float q = (b > 0) ? 
                  (-0.5 * (b + sqrtf(discr))) :
                  (-0.5 * (b - sqrtf(discr)));
        x0 = q/a;
        x1 = c/q;
    }
    if (x0 > x1)
    {
        float tmp = x0;
        x0 = x1;
        x1 = tmp;
    }
    return true;
}

/* This is a global CUDA function for controlling the calculation of intersection
 * times. It is a CUDA version of the visit function from ArrowIntersector.cc in
 * McVine (mcvine/packages/mccomposite/lib/geometry/visitors/ArrowIntersector.cc).
 */
__global__ void intersectBox(float* rx, float* ry, float* rz,
                             float* vx, float* vy, float* vz,
                             const float X, const float Y, const float Z, 
                             const int N, float* ts, float* pts)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        int offset = 0;
        if (vz[index] != 0)
        {
            intersectRectangle(ts, pts, rx[index], ry[index], rz[index], Z/2, vx[index], vy[index], vz[index], X, Y, 0, 6, 0, offset);
            intersectRectangle(ts, pts, rx[index], ry[index], rz[index], -Z/2, vx[index], vy[index], vz[index], X, Y, 0, 6, 1, offset);
        }
        else
        {
            ts[index*6] = -1;
            ts[index*6 + 1] = -1;
        }
        if (vx[index] != 0)
        {
            intersectRectangle(ts, pts, ry[index], rz[index], rx[index], X/2, vy[index], vz[index], vx[index], Y, Z, 1, 6, 2, offset);
            intersectRectangle(ts, pts, ry[index], rz[index], rx[index], -X/2, vy[index], vz[index], vx[index], Y, Z, 1, 6, 3, offset);
        }
        else
        {
            ts[index*6 + 2] = -1;
            ts[index*6 + 3] = -1;
        }
        if (vy[index] != 0)
        {
            intersectRectangle(ts, pts, rz[index], rx[index], ry[index], Y/2, vz[index], vx[index], vy[index], Z, X, 2, 6, 4, offset);
            intersectRectangle(ts, pts, rz[index], rx[index], ry[index], -Y/2, vz[index], vx[index], vy[index], Z, X, 2, 6, 5, offset);
        }
        else
        {
            ts[index*6 + 4] = -1;
            ts[index*6 + 5] = -1;
        }
    }
}

__global__ void intersectCylinder(float *rx, float *ry, float *rz,
                                  float *vx, float *vy, float *vz,
                                  const float r, const float h,
                                  const int N, float *ts, float *pts)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        int offset = 0;
        intersectCylinderTopBottom(ts, pts, rx[index], ry[index], rz[index], vx[index], vy[index], vz[index], r, h, offset);
        intersectCylinderSide(ts, pts, rx[index], ry[index], rz[index], vx[index], vy[index], vz[index], r, h, offset);
    }
}

__global__ void intersectPyramid(float *rx, float *ry, float *rz,
                                 float *vx, float *vy, float *vz,
                                 const float X, const float Y, const float H,
                                 const int N, float *ts, float *pts)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        int offset = 0;
        if (vz[index] != 0)
        {
            intersectRectangle(ts, pts, rx[index], ry[index], rz[index], -H, vx[index], vy[index], vz[index], X, Y, 0, 5, 0, offset);
        }
        intersectTriangle(ts, pts,
                          rx[index], ry[index], rz[index],
                          vz[index], vy[index], vz[index],
                          0, 0, 0, X/2, Y/2, -H, X/2, -Y/2, -H,
                          1, offset);
        intersectTriangle(ts, pts,
                          rx[index], ry[index], rz[index],
                          vz[index], vy[index], vz[index],
                          0, 0, 0, X/2, -Y/2, -H, -X/2, -Y/2, -H,
                          2, offset);
        intersectTriangle(ts, pts,
                          rx[index], ry[index], rz[index],
                          vz[index], vy[index], vz[index],
                          0, 0, 0, -X/2, -Y/2, -H, -X/2, Y/2, -H,
                          3, offset);
        intersectTriangle(ts, pts,
                          rx[index], ry[index], rz[index],
                          vz[index], vy[index], vz[index],
                          0, 0, 0, -X/2, Y/2, -H, X/2, Y/2, -H,
                          4, offset);
        //printf("index = %i:\n    ts[%i] = %f ts[%i] = %f ts[%i] = %f ts[%i] = %f ts[%i] = %f\n    rx[%i] = %f ry[%i] = %f rz[%i] = %f\n    vx[%i] = %f vy[%i] = %f vz[%i] = %f\n    pts[%i] = %f pts[%i] = %f pts[%i] = %f\n    pts[%i] = %f pts[%i] = %f pts[%i] = %f\n", index, 5*index, ts[5*index], 5*index+1, ts[5*index+1], 5*index+2, ts[5*index+2], 5*index+3, ts[5*index+3], 5*index+4, ts[5*index+4], index, rx[index], index, ry[index], index, rz[index], index, vx[index], index, vy[index], index, vz[index], 6*index, pts[6*index], 6*index+1, pts[6*index+1], 6*index+2, pts[6*index+2], 6*index+3, pts[6*index+3], 6*index+4, pts[6*index+4], 6*index+5, pts[6*index+5]);
    }
}

__global__ void intersectSphere(float *rx, float *ry, float *rz,
                                float *vx, float *vy, float *vz,
                                const float radius,
                                const int N, float *ts, float *pts)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        float a = dot(vx[index], vy[index], vz[index],
                      vx[index], vy[index], vz[index]);
        float b = 2 * dot(rx[index], ry[index], rz[index],
                          vx[index], vy[index], vz[index]);
        float c = dot(rx[index], ry[index], rz[index],
                      rx[index], ry[index], rz[index]);
        c -= radius*radius;
        float t0, t1;
        if (!solveQuadratic(a, b, c, t0, t1))
        {
            ts[2*index] = -1;
            ts[2*index + 1] = -1;
            return;
        }
        else
        {
            if (t0 < 0)
            {
                ts[2*index] = -1;
            }
            else
            {
                ts[2*index] = t0;
                pts[6*index] = rx[index] + vx[index] * t0;
                pts[6*index+1] = ry[index] + vy[index] * t0;
                pts[6*index+2] = rz[index] + vz[index] * t0;
            }
            if (t1 < 0)
            {
                ts[2*index+1] = -1;
            }
            else
            {
                ts[2*index + 1] = t1;
                pts[6*index+3] = rx[index] + vx[index] * t1;
                pts[6*index+4] = ry[index] + vy[index] * t1;
                pts[6*index+5] = rz[index] + vz[index] * t1;
            }
        }
        __syncthreads();
    }
}

__global__ void simplifyTimes(const float *times, const int N, const int groupSize, float *simp)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        int count = 0;
        for (int i = 0; i < groupSize; i++)
        {
            if (times[groupSize * index + i] != -1 && count < 2)
            {
                simp[2*index+count] = times[groupSize*index+i];
                count++;
            }
        }
    }
}

__global__ void prepRand(hiprandState *state, int seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(((seed << 10) + idx), 0, 0, &state[idx]); 
}

__device__ void randCoord(float* inters, float* time , float *sx, float *sy, float *sz, hiprandState *state)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float dt = time[1] - time[0];
    float mx = (inters[3] - inters[0])/dt;
    float my = (inters[4] - inters[1])/dt;
    float mz = (inters[5] - inters[2])/dt;
    float randt = hiprand_uniform(&(state[index]));
    randt *= dt;
    *sx = inters[0] + mx*randt;
    *sy = inters[1] + my*randt;
    *sz = inters[2] + mz*randt;
}

__global__ void calcScatteringSites(float* ts, float* int_pts, float* pos, hiprandState *state, const int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        if (ts[2*index] != -5 && ts[2*index+1] != -5)
        {
            if (ts[2*index] > ts[2*index+1])
            {
                float tmpt, tmpc;
                tmpt = ts[2*index];
                ts[2*index] = ts[2*index+1];
                ts[2*index+1] = tmpt;
                for (int i = 6*index; i < 6*index+3; i++)
                {
                    tmpc = int_pts[i];
                    int_pts[i] = int_pts[i + 3];
                    int_pts[i + 3] = tmpc;
                }
            }
            randCoord(&(int_pts[6*index]), &(ts[2*index]), &(pos[3*index + 0]), &(pos[3*index + 1]), &(pos[3*index + 2]), state);
        }
        /*else
        {
            pos[3*index + 0] = 20 * X;
            pos[3*index + 1] = 20 * Y;
            pos[3*index + 2] = 20 * Z;
        }*/
    }
}
