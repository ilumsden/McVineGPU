#include "hip/hip_runtime.h"
#include "UtilWrapper.hpp"

#include "Error.hpp"

namespace mcvine
{

    namespace gpu
    {

        namespace test
        {

            namespace kernels = mcvine::gpu::kernels;

            __global__ void testQuadratic(float *a, float *b, float *c,
                                          float *x0, float *x1,
                                          bool *solved, const int N)
            {
                int index = blockIdx.x * blockDim.x + threadIdx.x;
                if (index < N)
                {
                    solved[index] = kernels::solveQuadratic(a[index], b[index],
                                                            c[index], x0[index],
                                                            x1[index]);
                }
            }

            void testInitArray(std::vector<float> &data, const float val)
            {
                data.clear();
                data.resize(10);
                float *d_data;
                CudaErrchk( hipMalloc(&d_data, 10*sizeof(float)) );
                kernels::initArray<float><<<1, 10>>>(d_data, 10, val);
                CudaErrchkNoCode();
                float *d = data.data();
                CudaErrchk( hipMemcpy(d, d_data, 10*sizeof(float), hipMemcpyDeviceToHost) );
                CudaErrchk( hipFree(d_data) );
            }

            void testSolveQuadratic(float a, float b, float c,
                                    float &x0, float &x1, bool &solved)
            {
                float *d_a, *d_b, *d_c;
                float *d_x0, *d_x1;
                bool *d_solve;
                CudaErrchk( hipMalloc(&d_a, sizeof(float)) );
                CudaErrchk( hipMalloc(&d_b, sizeof(float)) );
                CudaErrchk( hipMalloc(&d_c, sizeof(float)) );
                CudaErrchk( hipMalloc(&d_x0, sizeof(float)) );
                CudaErrchk( hipMalloc(&d_x1, sizeof(float)) );
                CudaErrchk( hipMalloc(&d_solve, sizeof(bool)) );
                CudaErrchk( hipMemcpy(d_a, &a, sizeof(float), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_b, &b, sizeof(float), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_c, &c, sizeof(float), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_x0, &x0, sizeof(float), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_x1, &x1, sizeof(float), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_solve, &solved, sizeof(bool), hipMemcpyHostToDevice) );
                testQuadratic<<<1, 1>>>(d_a, d_b, d_c, d_x0, d_x1, d_solve, 1);
                CudaErrchkNoCode();
                CudaErrchk( hipMemcpy(&x0, d_x0, sizeof(float), hipMemcpyDeviceToHost) );
                CudaErrchk( hipMemcpy(&x1, d_x1, sizeof(float), hipMemcpyDeviceToHost) );
                CudaErrchk( hipMemcpy(&solved, d_solve, sizeof(bool), hipMemcpyDeviceToHost) );
                CudaErrchk( hipFree(d_a) );
                CudaErrchk( hipFree(d_b) );
                CudaErrchk( hipFree(d_c) );
                CudaErrchk( hipFree(d_x0) );
                CudaErrchk( hipFree(d_x1) );
                CudaErrchk( hipFree(d_solve) );
            }

            void testSimplifyPairs(std::vector<float> &times,
                                   std::vector< Vec3<float> > &coords, 
                                   const int input_groups,
                                   const int numOutputs)
            {
                // To simulate the behavior of the actual code, 
                // the coordinate groupsize is considered to be 2.
                int size_times = times.size();
                int size_coords = coords.size();
                int threads = size_coords / 2;
                float *d_times, *s_times;
                Vec3<float> *d_coords, *s_coords;
                CudaErrchk( hipMalloc(&d_times, size_times*sizeof(float)) );
                CudaErrchk( hipMalloc(&s_times, ((size_times*numOutputs)/input_groups)*sizeof(float)) );
                CudaErrchk( hipMalloc(&d_coords, size_coords*sizeof(Vec3<float>)) );
                CudaErrchk( hipMalloc(&s_coords, ((size_coords*numOutputs)/2)*sizeof(Vec3<float>)) );
                CudaErrchk( hipMemcpy(d_times, times.data(), size_times*sizeof(float), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_coords, coords.data(), size_coords*sizeof(Vec3<float>), hipMemcpyHostToDevice) );
                kernels::simplifyTimePointPairs<<<1, threads>>>(d_times, d_coords,
                                                                threads, 
                                                                input_groups, 2,
                                                                numOutputs,
                                                                s_times, s_coords);
                CudaErrchkNoCode();
                times.resize((size_times*numOutputs)/input_groups);
                coords.resize((size_coords*numOutputs)/2);
                float *dt = times.data();
                Vec3<float> *dc = coords.data();
                CudaErrchk( hipMemcpy(dt, s_times, ((size_times*numOutputs)/input_groups)*sizeof(float), hipMemcpyDeviceToHost) );
                CudaErrchk( hipMemcpy(dc, s_coords, ((size_coords*numOutputs)/2)*sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
                CudaErrchk( hipFree(d_times) );
                CudaErrchk( hipFree(s_times) );
                CudaErrchk( hipFree(d_coords) );
                CudaErrchk( hipFree(s_coords) );
            }

            void testForceIntOrder(std::vector<float> &ts,
                                   std::vector< Vec3<float> > &coords)
            {
                float *d_times;
                Vec3<float> *d_coords;
                CudaErrchk( hipMalloc(&d_times, (int)(ts.size())*sizeof(float)) );
                CudaErrchk( hipMalloc(&d_coords, (int)(coords.size())*sizeof(Vec3<float>)) );
                CudaErrchk( hipMemcpy(d_times, ts.data(), (int)(ts.size())*sizeof(float), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_coords, coords.data(), (int)(coords.size())*sizeof(Vec3<float>), hipMemcpyHostToDevice) );
                kernels::forceIntersectionOrder<<<1, ((int)(ts.size())/2)>>>(d_times, d_coords, ((int)(ts.size())/2));
                CudaErrchkNoCode();
                float *t = ts.data();
                Vec3<float> *dc = coords.data();
                CudaErrchk( hipMemcpy(t, d_times, (int)(ts.size())*sizeof(float), hipMemcpyDeviceToHost) );
                CudaErrchk( hipMemcpy(dc, d_coords, (int)(coords.size())*sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
                CudaErrchk( hipFree(d_times) );
                CudaErrchk( hipFree(d_coords) );
            }

            void testPropagate(Vec3<float> &orig, float &time,
                               Vec3<float> &new_orig, float &new_time)
            {
                Vec3<float> *d_orig, *d_pos;
                float *d_time, *d_ntime;
                CudaErrchk( hipMalloc(&d_orig, sizeof(Vec3<float>)) );
                CudaErrchk( hipMalloc(&d_pos, sizeof(Vec3<float>)) );
                CudaErrchk( hipMalloc(&d_time, sizeof(float)) );
                CudaErrchk( hipMalloc(&d_ntime, sizeof(float)) );
                CudaErrchk( hipMemcpy(d_orig, &orig, sizeof(Vec3<float>), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_pos, &new_orig, sizeof(Vec3<float>), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_time, &time, sizeof(float), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_ntime, &new_time, sizeof(float), hipMemcpyHostToDevice) );
                kernels::propagate<<<1, 1>>>(d_orig, d_time, d_pos, d_ntime, 1);
                CudaErrchkNoCode();
                CudaErrchk( hipMemcpy(&orig, d_orig, sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
                CudaErrchk( hipMemcpy(&time, d_time, sizeof(float), hipMemcpyDeviceToHost) );
                CudaErrchk( hipFree(d_orig) );
                CudaErrchk( hipFree(d_pos) );
                CudaErrchk( hipFree(d_time) );
                CudaErrchk( hipFree(d_ntime) );
            }

            void testUpdateProbability(float &prob,
                                       Vec3<float> &p1, Vec3<float> &p0)
            {
                float *d_prob;
                Vec3<float> *d_p1, *d_p0;
                CudaErrchk( hipMalloc(&d_prob, sizeof(float)) );
                CudaErrchk( hipMalloc(&d_p1, sizeof(Vec3<float>)) );
                CudaErrchk( hipMalloc(&d_p0, sizeof(Vec3<float>)) );
                CudaErrchk( hipMemcpy(d_prob, &prob, sizeof(float), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_p1, &p1, sizeof(Vec3<float>), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_p0, &p0, sizeof(Vec3<float>), hipMemcpyHostToDevice) );
                kernels::updateProbability<<<1, 1>>>(d_prob, d_p1, d_p0, 1, 1, atten, 1);
                CudaErrchkNoCode();
                CudaErrchk( hipMemcpy(&prob, d_prob, sizeof(float), hipMemcpyDeviceToHost) );
                CudaErrchk( hipFree(d_prob) );
                CudaErrchk( hipFree(d_p1) );
                CudaErrchk( hipFree(d_p0) );
            }

        }

    }

}
