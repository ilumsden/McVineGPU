#include "hip/hip_runtime.h"
#include "ScatteringWrapper.hpp"
#include "Error.hpp"

#include <cfloat>
#include <ctime>
#include <random>

namespace mcvine
{

    namespace gpu
    {

        namespace test
        {

            __global__ void testRandCoord(Vec3<float> *orig, Vec3<float> *vel,
                                          float *int_times, Vec3<float> *pos,
                                          float *scat_times, float *rands,
                                          const int N)
            {
                int index = blockIdx.x * blockDim.x + threadIdx.x;
                if (index < N)
                {
                    kernels::randCoord(orig[index], vel[index], &(int_times[2*index]), pos[index], scat_times[index], rands[index]);
                }
            }

            __global__ void testIsoScatterKernel(Vec3<float> *vel, float *rands,
                                                 const int N)
            {
                int index = blockIdx.x * blockDim.x + threadIdx.x;
                if (index < N)
                {
                    kernels::isotropicScatteringKernel(vel[index], &(rands[2*index]));
                }
            }

            void randTest(Vec3<float> &orig, Vec3<float> &vel,
                          float *int_times, Vec3<float> &pos,
                          float &scat_times)
            {
                pos = Vec3<float>(FLT_MAX, FLT_MAX, FLT_MAX);
                scat_times = -5;
                std::mt19937 rng(time(NULL));
                std::uniform_real_distribution<float> dist(0.f, 1.f);
                float r = dist(rng);
                Vec3<float> *d_orig, *d_vel, *d_pos;
                float *d_itimes, *d_stimes, *d_rand;
                CudaErrchk( hipMalloc(&d_orig, sizeof(Vec3<float>)) );
                CudaErrchk( hipMalloc(&d_vel, sizeof(Vec3<float>)) );
                CudaErrchk( hipMalloc(&d_pos, sizeof(Vec3<float>)) );
                CudaErrchk( hipMalloc(&d_itimes, 2*sizeof(float)) );
                CudaErrchk( hipMalloc(&d_stimes, sizeof(float)) );
                CudaErrchk( hipMalloc(&d_rand, sizeof(float)) );
                CudaErrchk( hipMemcpy(d_orig, &orig, sizeof(Vec3<float>), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_vel, &vel, sizeof(Vec3<float>), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_pos, &pos, sizeof(Vec3<float>), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_itimes, int_times, 2*sizeof(float), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_stimes, &scat_times, sizeof(float), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_rand, &r, sizeof(float), hipMemcpyHostToDevice) );
                testRandCoord<<<1, 1>>>(d_orig, d_vel, d_itimes, d_pos, d_stimes, d_rand, 1);
                CudaErrchk( hipMemcpy(&pos, d_pos, sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
                CudaErrchk( hipMemcpy(&scat_times, d_stimes, sizeof(float), hipMemcpyDeviceToHost) );
            }

            void scatteringSiteTest(Vec3<float> &orig, Vec3<float> &vel, 
                                    float *int_times, Vec3<float> &pos,
                                    float &scat_times)
            {
                pos = Vec3<float>(FLT_MAX, FLT_MAX, FLT_MAX);
                scat_times = -5;
                std::mt19937 rng(time(NULL));
                std::uniform_real_distribution<float> dist(0.f, 1.f);
                float r = dist(rng);
                Vec3<float> *d_orig, *d_vel, *d_pos;
                float *d_itimes, *d_stimes, *d_rand;
                CudaErrchk( hipMalloc(&d_orig, sizeof(Vec3<float>)) );
                CudaErrchk( hipMalloc(&d_vel, sizeof(Vec3<float>)) );
                CudaErrchk( hipMalloc(&d_pos, sizeof(Vec3<float>)) );
                CudaErrchk( hipMalloc(&d_itimes, 2*sizeof(float)) );
                CudaErrchk( hipMalloc(&d_stimes, sizeof(float)) );
                CudaErrchk( hipMalloc(&d_rand, sizeof(float)) );
                CudaErrchk( hipMemcpy(d_orig, &orig, sizeof(Vec3<float>), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_vel, &vel, sizeof(Vec3<float>), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_pos, &pos, sizeof(Vec3<float>), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_itimes, int_times, 2*sizeof(float), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_stimes, &scat_times, sizeof(float), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_rand, &r, sizeof(float), hipMemcpyHostToDevice) );
                kernels::calcScatteringSites<<<1, 1>>>(d_itimes, d_orig, d_vel, d_pos, d_stimes, d_rand, 1);
                CudaErrchk( hipMemcpy(&pos, d_pos, sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
                CudaErrchk( hipMemcpy(&scat_times, d_stimes, sizeof(float), hipMemcpyDeviceToHost) );
            }

            void isoScatterTest(Vec3<float> &vel)
            {
                float *rands = new float[2];
                std::mt19937 rng(time(NULL));
                std::uniform_real_distribution<float> dist(0.f, 1.f);
                rands[0] = dist(rng);
                rands[1] = dist(rng);
                Vec3<float> *d_vel;
                float *d_rand;
                CudaErrchk( hipMalloc(&d_vel, sizeof(Vec3<float>)) );
                CudaErrchk( hipMalloc(&d_rand, 2*sizeof(float)) );
                CudaErrchk( hipMemcpy(d_vel, &vel, sizeof(Vec3<float>), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_rand, rands, 2*sizeof(float), hipMemcpyHostToDevice) );
                testIsoScatterKernel<<<1, 1>>>(d_vel, d_rand, 1);
                CudaErrchk( hipMemcpy(&vel, d_vel, sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
                delete [] rands;
            }

            void scatterTest(const int key, const float &time,
                             Vec3<float> &vel)
            {
                float *rands = new float[2];
                //std::mt19937 rng(time(NULL));
                std::mt19937 rng;
                std::uniform_real_distribution<float> dist(0.f, 1.f);
                rands[0] = dist(rng);
                rands[1] = dist(rng);
                Vec3<float> *d_vel;
                float *d_rand, *d_times;
                CudaErrchk( hipMalloc(&d_vel, sizeof(Vec3<float>)) );
                CudaErrchk( hipMalloc(&d_rand, 2*sizeof(float)) );
                CudaErrchk( hipMalloc(&d_times, sizeof(float)) );
                CudaErrchk( hipMemcpy(d_vel, &vel, sizeof(Vec3<float>), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_rand, rands, 2*sizeof(float), hipMemcpyHostToDevice) );
                CudaErrchk( hipMemcpy(d_times, &time, sizeof(float), hipMemcpyHostToDevice) );
                kernels::scatter<<<1, 1>>>(key, d_times, d_vel, d_rand, 1);
                CudaErrchkNoCode();
                CudaErrchk( hipMemcpy(&vel, d_vel, sizeof(Vec3<float>), hipMemcpyDeviceToHost) );
                delete [] rands;
            }

        }

    }

}
